#include "poseidon.cuh"

/// These are pre-calculated constants for different curves
#if CURVE_ID == BN254
#include "appUtils/poseidon/constants/bn254_poseidon.h"
using namespace poseidon_constants_bn254;
#elif CURVE_ID == BLS12_381
#include "appUtils/poseidon/constants/bls12_381_poseidon.h"
using namespace poseidon_constants_bls12_381;
#elif CURVE_ID == BLS12_377
#include "appUtils/poseidon/constants/bls12_377_poseidon.h"
using namespace poseidon_constants_bls12_377;
#elif CURVE_ID == BW6_761
#include "appUtils/poseidon/constants/bw6_761_poseidon.h"
using namespace poseidon_constants_bw6_761;
#elif CURVE_ID == GL
#include "appUtils/poseidon/constants/goldilocks_poseidon.h"
using namespace poseidon_constants_goldilocks;
#endif

namespace poseidon {
#if CURVE_ID != GL
  template <typename S>
  hipError_t create_optimized_poseidon_constants(
    int arity,
    int full_rounds_half,
    int partial_rounds,
    const S* constants,
    device_context::DeviceContext& ctx,
    PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = ctx.stream;
    int width = arity + 1;
    int round_constants_len = width * full_rounds_half * 2 + partial_rounds;
    int mds_matrix_len = width * width;
    int sparse_matrices_len = (width * 2 - 1) * partial_rounds;
    int constants_len = round_constants_len + mds_matrix_len * 2 + sparse_matrices_len;

    // Malloc memory for copying constants
    S* d_constants;
    CHK_IF_RETURN(hipMallocAsync(&d_constants, sizeof(S) * constants_len, stream));

    // Copy constants
    CHK_IF_RETURN(hipMemcpyAsync(d_constants, constants, sizeof(S) * constants_len, hipMemcpyHostToDevice, stream));

    S* round_constants = d_constants;
    S* mds_matrix = round_constants + round_constants_len;
    S* non_sparse_matrix = mds_matrix + mds_matrix_len;
    S* sparse_matrices = non_sparse_matrix + mds_matrix_len;

    // Pick the domain_tag accordinaly
    // For now, we only support Merkle tree mode
    uint32_t tree_domain_tag_value = 1;
    tree_domain_tag_value = (tree_domain_tag_value << (width - 1)) - tree_domain_tag_value;
    S domain_tag = S::from(tree_domain_tag_value);

    // Make sure all the constants have been copied
    CHK_IF_RETURN(hipStreamSynchronize(stream));
    *poseidon_constants = {arity,      partial_rounds,    full_rounds_half, round_constants,
                           mds_matrix, non_sparse_matrix, sparse_matrices,  domain_tag};

    return CHK_LAST();
  }

  template <typename S>
  hipError_t init_optimized_poseidon_constants(
    int arity, device_context::DeviceContext& ctx, PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    int full_rounds_half = FULL_ROUNDS_DEFAULT;
    int partial_rounds;
    unsigned char* constants;
    switch (arity) {
    case 2:
      constants = poseidon_constants_2;
      partial_rounds = partial_rounds_2;
      break;
    case 4:
      constants = poseidon_constants_4;
      partial_rounds = partial_rounds_4;
      break;
    case 8:
      constants = poseidon_constants_8;
      partial_rounds = partial_rounds_8;
      break;
    case 11:
      constants = poseidon_constants_11;
      partial_rounds = partial_rounds_11;
      break;
    default:
      THROW_ICICLE_ERR(
        IcicleError_t::InvalidArgument, "init_optimized_poseidon_constants: #arity must be one of [2, 4, 8, 11]");
    }
    S* h_constants = reinterpret_cast<S*>(constants);

    create_optimized_poseidon_constants(arity, full_rounds_half, partial_rounds, h_constants, ctx, poseidon_constants);

    return CHK_LAST();
  }
#else
  template <typename S>
  hipError_t create_optimized_poseidon_constants(
    int arity,
    int full_rounds_half,
    int partial_rounds,
    const S* constants,
    int h_mds0to0,
    device_context::DeviceContext& ctx,
    PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = ctx.stream;
    int all_round_constants_len = 360;
    int mds_matrix_circ_len = 12;
    int mds_matrix_diag_len = 12;
    int fast_partial_first_round_constant_len = 12;
    int fast_partial_round_initial_matrix_len_x = 11;
    int fast_partial_round_initial_matrix_len_y = 11;
    int fast_partial_round_constants_len = 22;
    int fast_partial_round_w_hats_len_x = 22;
    int fast_partial_round_w_hats_len_y = 11;
    int fast_partial_round_vs_len_x = 22;
    int fast_partial_round_vs_len_y = 11;
    int constants_len =
      all_round_constants_len + mds_matrix_circ_len + mds_matrix_diag_len + fast_partial_first_round_constant_len +
      fast_partial_round_initial_matrix_len_x * fast_partial_round_initial_matrix_len_y +
      fast_partial_round_constants_len + fast_partial_round_w_hats_len_x * fast_partial_round_w_hats_len_y +
      fast_partial_round_vs_len_x * fast_partial_round_vs_len_y + 1;

    // Malloc memory for copying constants
    S* d_constants;
    CHK_IF_RETURN(hipMallocAsync(&d_constants, sizeof(S) * constants_len, stream));

    // Copy constants
    CHK_IF_RETURN(hipMemcpyAsync(d_constants, constants, sizeof(S) * constants_len, hipMemcpyHostToDevice, stream));

    S* all_round_constants = d_constants;
    S* mds_matrix_circ = all_round_constants + all_round_constants_len;
    S* mds_matrix_diag = mds_matrix_circ + mds_matrix_circ_len;
    S* fast_partial_first_round_constant = mds_matrix_diag + mds_matrix_diag_len;
    S* fast_partial_round_initial_matrix;
    fast_partial_round_initial_matrix = fast_partial_first_round_constant + fast_partial_first_round_constant_len;
    S* fast_partial_round_constants = fast_partial_round_initial_matrix +
                                      fast_partial_round_initial_matrix_len_x * fast_partial_round_initial_matrix_len_y;
    S* fast_partial_round_w_hats;
    fast_partial_round_w_hats = fast_partial_round_constants + fast_partial_round_constants_len;
    S* fast_partial_round_vs;
    fast_partial_round_vs =
      fast_partial_round_w_hats + fast_partial_round_w_hats_len_x * fast_partial_round_w_hats_len_y;

    // Make sure all the constants have been copied
    CHK_IF_RETURN(hipStreamSynchronize(stream));
    *poseidon_constants = {
      arity,
      partial_rounds,
      full_rounds_half,
      all_round_constants,
      mds_matrix_circ,
      mds_matrix_diag,
      fast_partial_first_round_constant,
      fast_partial_round_initial_matrix,
      fast_partial_round_constants,
      fast_partial_round_w_hats,
      fast_partial_round_vs,
      h_mds0to0,
      fast_partial_round_initial_matrix_len_y,
      fast_partial_round_w_hats_len_y,
      fast_partial_round_vs_len_y};

    return CHK_LAST();
  }

  template <typename S>
  hipError_t init_optimized_poseidon_constants(
    int arity, device_context::DeviceContext& ctx, PoseidonConstants<S>* poseidon_constants)
  {
    CHK_INIT_IF_RETURN();
    int full_rounds_half = 4;
    int partial_rounds;
    unsigned char* constants;
    switch (arity) {
    case 11:
      constants = poseidon_constants_11;
      partial_rounds = partial_rounds_11;
      break;
    default:
      THROW_ICICLE_ERR(IcicleError_t::InvalidArgument, "init_optimized_poseidon_constants: #arity must be 11");
    }
    S* h_constants = reinterpret_cast<S*>(constants);
    printf("create_optimized_poseidon_constants\n");
    create_optimized_poseidon_constants(
      arity, full_rounds_half, partial_rounds, h_constants, mds0to0, ctx, poseidon_constants);
    printf("create_optimized_poseidon_constants done\n");
    return CHK_LAST();
  }
#endif

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CreateOptimizedPoseidonConstants)(
    int arity,
    int full_rounds_half,
    int partial_rounds,
    const curve_config::scalar_t* constants,
    device_context::DeviceContext& ctx,
    PoseidonConstants<curve_config::scalar_t>* poseidon_constants)
  {
    return create_optimized_poseidon_constants<curve_config::scalar_t>(
      arity, full_rounds_half, partial_rounds, constants, mds0to0, ctx, poseidon_constants);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, InitOptimizedPoseidonConstants)(
    int arity, device_context::DeviceContext& ctx, PoseidonConstants<curve_config::scalar_t>* constants)
  {
    return init_optimized_poseidon_constants<curve_config::scalar_t>(arity, ctx, constants);
  }
} // namespace poseidon