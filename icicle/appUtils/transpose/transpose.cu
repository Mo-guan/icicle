#include "transpose.cuh"
#include "kernels.cu"

namespace transpose {

  template <typename S>
  hipError_t transpose_mem(S* input, S* output, const TransposeConfig& config)
  {
    CHK_INIT_IF_RETURN();
    hipStream_t& stream = config.ctx.stream;
    S* src;
    if (config.are_inputs_on_device) {
      src = input;
    } else {
      printf("matrix size in bytes: %lu\n", TKC<S>::size_bytes(config));
      CHK_IF_RETURN(hipMallocAsync(&src, TKC<S>::size_bytes(config), stream))
      CHK_IF_RETURN(hipMemcpyAsync(src, input, TKC<S>::size_bytes(config), hipMemcpyHostToDevice, stream));
    }

    S* dst;
    if (config.are_outputs_on_device) {
      dst = output;
    } else {
      CHK_IF_RETURN(hipMallocAsync(&dst, TKC<S>::size_bytes(config), stream))
    }

    dim3 grid = TKC<S>::get_grid(config);
    dim3 block = TKC<S>::get_block();

    // transposeSmemUnrollPadDynP<<<grid, block, 0, stream>>>();
    transposeSmemUnrollPadDyn<S><<<grid, block, TKC<S>::sm_size_bytes(), stream>>>(src, dst, config.nrows, config.ncols);
    // checkaa<8><<<grid, block>>>();

    if (!config.are_inputs_on_device) CHK_IF_RETURN(hipFreeAsync(src, stream));

    if (!config.are_outputs_on_device) {
      CHK_IF_RETURN(hipMemcpyAsync(output, dst, TKC<S>::size_bytes(config), hipMemcpyDeviceToHost, stream));
      CHK_IF_RETURN(hipFreeAsync(dst, stream));
    }

    if (!config.is_async) return CHK_STICKY(hipStreamSynchronize(stream));
    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, TransposeMem)(
    curve_config::scalar_t* input, curve_config::scalar_t* output, TransposeConfig& config)
  {
    transpose_mem<curve_config::scalar_t>(input, output, config);
    return CHK_LAST();
  }
} // namespace transpose