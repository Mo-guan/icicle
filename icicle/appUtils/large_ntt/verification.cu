#include "hip/hip_runtime.h"

#define CURVE_ID 1 // TODO Yuval: move to makefile

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>

#include "curves/curve_config.cuh"
#include "ntt/ntt.cu"
#include "large_ntt/large_ntt.cuh"
#include <memory>

#define PERFORMANCE

typedef curve_config::scalar_t test_scalar;
#include "kernel_ntt.cu"

#define $CUDA(call)                                                                                                    \
  if ((call) != 0) {                                                                                                   \
    printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError());        \
    exit(1);                                                                                                           \
  }

void random_samples(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_scalar::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i ? res[i - 1] + test_scalar::one() * test_scalar::omega(4) : test_scalar::zero();
}

int main(int argc, char** argv)
{
#ifdef PERFORMANCE
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float icicle_time, new_time;
#endif

  int NTT_LOG_SIZE = (argc > 1) ? atoi(argv[1]) : 22; // assuming second input is the log-size
  int NTT_SIZE = 1 << NTT_LOG_SIZE;
  int INV = false;
  const ntt::Ordering ordering = ntt::Ordering::kNN;
  const char* ordering_str = ordering == ntt::Ordering::kNN   ? "NN"
                             : ordering == ntt::Ordering::kNR ? "NR"
                             : ordering == ntt::Ordering::kRN ? "RN"
                                                              : "RR";

  printf("running ntt 2^%d, INV=%d, ordering=%s\n", NTT_LOG_SIZE, INV, ordering_str);

  // cpu allocation
  auto CpuScalars = std::make_unique<test_scalar[]>(NTT_SIZE);
  auto CpuOutputOld = std::make_unique<test_scalar[]>(NTT_SIZE);
  auto CpuOutputNew = std::make_unique<test_scalar[]>(NTT_SIZE);

  // gpu allocation
  test_scalar *GpuScalars, *GpuOutputOld, *GpuOutputNew;
  $CUDA(hipMalloc(&GpuScalars, sizeof(test_scalar) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputOld, sizeof(test_scalar) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputNew, sizeof(test_scalar) * NTT_SIZE));

  // init inputs
  random_samples(CpuScalars.get(), NTT_SIZE);
  $CUDA(hipMemcpy(GpuScalars, CpuScalars.get(), NTT_SIZE, hipMemcpyHostToDevice));

  // init
  auto ntt_config = ntt::DefaultNTTConfig<test_scalar>();
  ntt_config.ordering = ordering;
  ntt_config.are_inputs_on_device = true;
  ntt_config.are_outputs_on_device = true;
  // ntt_config.is_async = true;

  const test_scalar basic_root = test_scalar::omega(NTT_LOG_SIZE);
  ntt::InitDomain(basic_root, ntt_config.ctx);

  $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));

#ifdef PERFORMANCE
  $CUDA(hipEventCreate(&icicle_start));
  $CUDA(hipEventCreate(&icicle_stop));
  $CUDA(hipEventCreate(&new_start));
  $CUDA(hipEventCreate(&new_stop));

  // run ntt
  auto benchmark = [&](bool is_print, int iterations) {
    // NEW
    $CUDA(hipEventRecord(new_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = false; // mixed-radix ntt (a.k.a new ntt)
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputNew);
    }
    $CUDA(hipEventRecord(new_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&new_time, new_start, new_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    // OLD
    $CUDA(hipEventRecord(icicle_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = true;
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
    }
    $CUDA(hipEventRecord(icicle_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    if (is_print) {
      printf("Old Runtime=%0.3f MS\n", icicle_time / iterations);
      printf("New Runtime=%0.3f MS\n", new_time / iterations);
    }
  };

  int count = 1;
  benchmark(false /*=print*/, 1); // warmup - is this applicable to real usecase??
  benchmark(true /*=print*/, count);
#else
  ntt_config.is_force_radix2 = false;
  ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputNew);
  printf("finished new\n");

  ntt_config.is_force_radix2 = true;
  ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
  printf("finished old\n");

  // verify
  $CUDA(hipMemcpy(CpuOutputNew.get(), GpuOutputNew, NTT_SIZE, hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(CpuOutputOld.get(), GpuOutputOld, NTT_SIZE, hipMemcpyDeviceToHost));
#endif // PERFORMANCE

  bool success = true;
  for (int i = 0; i < NTT_SIZE; i++) {
    if (CpuOutputNew[i] != CpuOutputOld[i]) {
      success = false;
      std::cout << i << " ref " << CpuOutputOld[i] << " != " << CpuOutputNew[i] << std::endl;
      // break;
    } else {
      // std::cout << i << " ref " << CpuOutputOld[i] << " == " << CpuOutputNew[i] << std::endl;
      // break;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  fprintf(stderr, "%s\n", success_str);

  $CUDA(hipFree(GpuScalars));
  $CUDA(hipFree(GpuOutputOld));
  $CUDA(hipFree(GpuOutputNew));

  return 0;
}