#include "hip/hip_runtime.h"

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>
#include "../../curves/bls12_377/curve_config.cuh"
#include "ntt.cuh"
// #include "../../curves/bn254/curve_config.cuh"

// #include <stdio.h>
// #include <stdint.h>
// #include <hip/hip_cooperative_groups.h>

// #define PERFORMANCE

using namespace BLS12_377;
typedef scalar_t test_scalar;
#include "kernel_ntt.cu"

#define $CUDA(call) if((call)!=0) { printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError()); exit(1); }

void random_samples(test_scalar* res, uint32_t count) {
  for(int i=0;i<count;i++)
    // res[i]= i<1000? test_scalar::rand_host() : res[i-1000];
    res[i]= i==64? test_scalar::one() : test_scalar::zero();
}

void incremental_values(test_scalar* res, uint32_t count) {
  for(int i=0;i<count;i++)
    res[i]=i? res[i-1]+test_scalar::one() : test_scalar::zero();
}

int main(){

  #ifdef PERFORMANCE
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float       icicle_time, new_time;
  #endif

  int NTT_LOG_SIZE = 18;
  int TT_LOG_SIZE = 24;
  int NTT_SIZE = 1<<NTT_LOG_SIZE;
  int TT_SIZE = 1<<TT_LOG_SIZE;

  //cpu allocation
  test_scalar* cpuIcicle;
  uint4* cpuNew;
  uint4* cpuNew2;
  cpuIcicle=(test_scalar*)malloc(sizeof(test_scalar)*NTT_SIZE);
  cpuNew=(uint4*)malloc(sizeof(uint4)*NTT_SIZE*2);
  cpuNew2=(uint4*)malloc(sizeof(uint4)*NTT_SIZE*2);
  if(cpuIcicle==NULL || cpuNew==NULL || cpuNew2==NULL) {
    fprintf(stderr, "Malloc failed\n");
    exit(1);
  }

  //gpu allocation
  test_scalar* gpuIcicle;
  uint4* gpuNew;
  uint4* gpuNew2;
  uint4* gpuTwiddles;
  uint4* gpuIntTwiddles;
  $CUDA(hipMalloc((void**)&gpuIcicle, sizeof(test_scalar)*NTT_SIZE));
  $CUDA(hipMalloc((void**)&gpuNew, sizeof(uint4)*NTT_SIZE*2));
  $CUDA(hipMalloc((void**)&gpuNew2, sizeof(uint4)*NTT_SIZE*2));
  $CUDA(hipMalloc((void**)&gpuTwiddles, sizeof(uint4)*TT_SIZE*2));
  // $CUDA(hipMalloc((void**)&gpuIntTwiddles, sizeof(uint4)*TT_SIZE*2));

  //init inputs
  random_samples(cpuIcicle, NTT_SIZE);
  // incremental_values(cpuIcicle, NTT_SIZE);
  for (int i = 0; i < NTT_SIZE; i++)
  {
    cpuNew[i] = cpuIcicle[i].load_half(false);
    cpuNew[NTT_SIZE + i] = cpuIcicle[i].load_half(true);
    cpuNew2[i] = uint4{0,0,0,0};
    cpuNew2[NTT_SIZE + i] = uint4{0,0,0,0};
  }
  $CUDA(hipMemcpy(gpuIcicle, cpuIcicle, sizeof(test_scalar)*NTT_SIZE, hipMemcpyHostToDevice));
  $CUDA(hipMemcpy(gpuNew, cpuNew, sizeof(uint4)*NTT_SIZE*2, hipMemcpyHostToDevice));
  $CUDA(hipMemcpy(gpuNew2, cpuNew2, sizeof(uint4)*NTT_SIZE*2, hipMemcpyHostToDevice));
  gpuIntTwiddles = generate_external_twiddles(gpuTwiddles, TT_LOG_SIZE);
  // generate_internal_twiddles<<<1,1>>>(gpuIntTwiddles);
  hipDeviceSynchronize();
  printf("cuda err %d\n",hipGetLastError());

  #ifdef PERFORMANCE
  $CUDA(hipEventCreate(&icicle_start));
  $CUDA(hipEventCreate(&icicle_stop));
  $CUDA(hipEventCreate(&new_start));
  $CUDA(hipEventCreate(&new_stop));
  
  

  //run ntts
  int count = 10000;
  $CUDA(hipEventRecord(new_start, 0));
  // ntt64<<<1, 8, 512*sizeof(uint4)>>>(gpuNew, gpuNew, gpuTwiddles, NTT_LOG_SIZE ,1,0);
  for (size_t i = 0; i < count; i++)
    new_ntt(gpuNew, gpuNew2, gpuTwiddles, gpuIntTwiddles, NTT_LOG_SIZE, TT_LOG_SIZE);
    // new_ntt(gpuNew, gpuNew2, gpuTwiddles, NTT_LOG_SIZE);
  $CUDA(hipEventRecord(new_stop, 0));
  $CUDA(hipDeviceSynchronize());
  $CUDA(hipEventElapsedTime(&new_time, new_start, new_stop));
  hipDeviceSynchronize();
  printf("cuda err %d\n",hipGetLastError());
  test_scalar *icicle_tw;
  icicle_tw = fill_twiddle_factors_array(NTT_SIZE, test_scalar::omega(NTT_LOG_SIZE), 0);
  $CUDA(hipEventRecord(icicle_start, 0));
  for (size_t i = 0; i < count; i++)
    ntt_inplace_batch_template<test_scalar, test_scalar>(gpuIcicle, icicle_tw, NTT_SIZE, 1, false, false, nullptr, 0, false);
  $CUDA(hipEventRecord(icicle_stop, 0));
  $CUDA(hipDeviceSynchronize());
  $CUDA(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
  hipDeviceSynchronize();
  printf("cuda err %d\n",hipGetLastError());
  fprintf(stderr, "Icicle Runtime=%0.3f MS\n", icicle_time);
  fprintf(stderr, "New Runtime=%0.3f MS\n", new_time);
  #else
  new_ntt(gpuNew, gpuNew2, gpuTwiddles, gpuIntTwiddles, NTT_LOG_SIZE, TT_LOG_SIZE);
  reorder64_kernel<<<(1<<(NTT_LOG_SIZE-6)),64>>>(gpuNew, gpuNew2, NTT_SIZE/64);
  // new_ntt(gpuNew, gpuNew2, gpuTwiddles, NTT_LOG_SIZE);
  ntt_end2end_batch_template<test_scalar, test_scalar>(gpuIcicle, NTT_SIZE, NTT_SIZE, false, 0);
  reverse_order_batch(gpuIcicle, NTT_SIZE, NTT_LOG_SIZE, 1, 0);
  
  //verify
  $CUDA(hipMemcpy(cpuIcicle, gpuIcicle, sizeof(test_scalar)*NTT_SIZE, hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(cpuNew, gpuNew, sizeof(uint4)*NTT_SIZE*2, hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(cpuNew2, gpuNew2, sizeof(uint4)*NTT_SIZE*2, hipMemcpyDeviceToHost));
  // for (int i = 0; i < NTT_SIZE; i++)
  // {
  //   test_scalar new_temp;
  //   new_temp.store_half(cpuNew[i], false);
  //   new_temp.store_half(cpuNew[i+NTT_SIZE], true);
  //   if (i%64 == 0) printf("%d\n",i/64);
  //   std::cout << new_temp <<std::endl;
  // }
  // printf("\n\n");

  bool success = true;
  for (int i = 0; i < NTT_SIZE; i++)
  {
    if (i%(64*64) >= 64*2) continue;
    test_scalar icicle_temp, new_temp;
    icicle_temp = cpuIcicle[i];
    new_temp.store_half(cpuNew[i], false);
    new_temp.store_half(cpuNew[i+NTT_SIZE], true);
    if (i%64 == 0) printf("%d\n",i/64);
    if (icicle_temp != new_temp){
      success = false;
      std::cout << "ref "<< icicle_temp << " != " << new_temp <<std::endl;
    }
    else{
      std::cout << "ref "<< icicle_temp << " == " << new_temp <<std::endl;
    }
  }
  if (success){
    printf("success!\n");
  }
  #endif

  return 0;

}