#include "hip/hip_runtime.h"

#define CURVE_ID BLS12_381

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>

#include "curves/curve_config.cuh"
#include "ntt/ntt.cu"
#include "ntt/ntt_impl.cuh"
#include <memory>

typedef curve_config::scalar_t test_scalar;
typedef curve_config::scalar_t test_data; // uncomment for NTT
#include "kernel_ntt.cu"

#define $CUDA(call)                                                                                                    \
  if ((call) != 0) {                                                                                                   \
    printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError());        \
    exit(1);                                                                                                           \
  }

void random_samples(test_data* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_data::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++) {
    res[i] = i ? res[i - 1] + test_scalar::one() : test_scalar::zero();
  }
}

int main(int argc, char** argv)
{
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float icicle_time, new_time;

  int NTT_LOG_SIZE = (argc > 1) ? atoi(argv[1]) : 19; // assuming second input is the log-size
  int NTT_SIZE = 1 << NTT_LOG_SIZE;
  bool INPLACE = (argc > 2) ? atoi(argv[2]) : false;
  int INV = (argc > 3) ? atoi(argv[3]) : false;
  int BATCH_SIZE = (argc > 4) ? atoi(argv[4]) : 1<<5;

  const ntt::Ordering ordering = ntt::Ordering::kNN;
  const char* ordering_str = ordering == ntt::Ordering::kNN   ? "NN"
                             : ordering == ntt::Ordering::kNR ? "NR"
                             : ordering == ntt::Ordering::kRN ? "RN"
                                                              : "RR";

  printf(
    "running ntt 2^%d, batch_size=%d, ordering=%s, inplace=%d, inverse=%d\n", NTT_LOG_SIZE, BATCH_SIZE, ordering_str,
    INPLACE, INV);

  $CUDA(hipFree(nullptr)); // init GPU context (warmup)

  // init domain
  auto ntt_config = ntt::DefaultNTTConfig<test_scalar>();
  ntt_config.ordering = ordering;
  ntt_config.are_inputs_on_device = true;
  ntt_config.are_outputs_on_device = true;
  ntt_config.batch_size = BATCH_SIZE;

  $CUDA(hipEventCreate(&icicle_start));
  $CUDA(hipEventCreate(&icicle_stop));
  $CUDA(hipEventCreate(&new_start));
  $CUDA(hipEventCreate(&new_stop));

  auto start = std::chrono::high_resolution_clock::now();
  const test_scalar basic_root = test_scalar::omega(NTT_LOG_SIZE);
  ntt::InitDomain(basic_root, ntt_config.ctx);
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
  std::cout << "initDomain took: " << duration / 1000 << " MS" << std::endl;

  // cpu allocation
  auto CpuScalars = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);
  auto CpuOutputOld = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);
  auto CpuOutputNew = std::make_unique<test_data[]>(NTT_SIZE * BATCH_SIZE);

  // gpu allocation
  test_data *GpuScalars, *GpuOutputOld, *GpuOutputNew;
  $CUDA(hipMalloc(&GpuScalars, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));
  $CUDA(hipMalloc(&GpuOutputOld, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));
  $CUDA(hipMalloc(&GpuOutputNew, sizeof(test_data) * NTT_SIZE * BATCH_SIZE));

  // init inputs
  // incremental_values(CpuScalars.get(), NTT_SIZE * BATCH_SIZE);
  random_samples(CpuScalars.get(), NTT_SIZE * BATCH_SIZE);
  $CUDA(hipMemcpy(GpuScalars, CpuScalars.get(), NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyHostToDevice));

  // inplace
  if (INPLACE) {
    $CUDA(hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice));
  }

  // run ntt
  auto benchmark = [&](bool is_print, int iterations) {
    // NEW
    $CUDA(hipEventRecord(new_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = false; // mixed-radix ntt (a.k.a new ntt)
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(
        INPLACE ? GpuOutputNew : GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config,
        GpuOutputNew);
    }
    $CUDA(hipEventRecord(new_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&new_time, new_start, new_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    // OLD
    $CUDA(hipEventRecord(icicle_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = true;
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
    }
    $CUDA(hipEventRecord(icicle_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    if (is_print) {
      printf("Old Runtime=%0.3f MS\n", icicle_time / iterations);
      printf("New Runtime=%0.3f MS\n", new_time / iterations);
    }
  };

  benchmark(false /*=print*/, 1); // warmup
  int count = INPLACE ? 1 : 10;
  if (INPLACE) {
    $CUDA(hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice));
  }
  benchmark(true /*=print*/, count);

  // verify
  $CUDA(
    hipMemcpy(CpuOutputNew.get(), GpuOutputNew, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));
  $CUDA(
    hipMemcpy(CpuOutputOld.get(), GpuOutputOld, NTT_SIZE * BATCH_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));

  bool success = true;
  for (int i = 0; i < NTT_SIZE * BATCH_SIZE; i++) {
    if (CpuOutputNew[i] != CpuOutputOld[i]) {
      success = false;
      // std::cout << i << " ref " << CpuOutputOld[i] << " != " << CpuOutputNew[i] << std::endl;
      break;
    } else {
      // std::cout << i << " ref " << CpuOutputOld[i] << " == " << CpuOutputNew[i] << std::endl;
      // break;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  printf("%s\n", success_str);

  $CUDA(hipFree(GpuScalars));
  $CUDA(hipFree(GpuOutputOld));
  $CUDA(hipFree(GpuOutputNew));

  return 0;
}