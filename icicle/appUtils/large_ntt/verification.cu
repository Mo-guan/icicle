#include "hip/hip_runtime.h"

#define CURVE_ID BLS12_381

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>

#include "curves/curve_config.cuh"
#include "ntt/ntt.cu"
#include "ntt/ntt_impl.cuh"
#include <memory>

#define PERFORMANCE

typedef curve_config::scalar_t test_scalar;
typedef curve_config::scalar_t test_data; // uncomment for NTT
// typedef curve_config::projective_t test_data; // uncomment for ECNTT
#include "kernel_ntt.cu"

#define $CUDA(call)                                                                                                    \
  if ((call) != 0) {                                                                                                   \
    printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError());        \
    exit(1);                                                                                                           \
  }

void random_samples(test_data* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_data::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++) {
    res[i] = i ? res[i - 1] + test_scalar::one() * test_scalar::omega(4) : test_scalar::zero();
  }
}

int main(int argc, char** argv)
{
#ifdef PERFORMANCE
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float icicle_time, new_time;
#endif

  int NTT_LOG_SIZE = (argc > 1) ? atoi(argv[1]) : 19; // assuming second input is the log-size
  int NTT_SIZE = 1 << NTT_LOG_SIZE;
  bool INPLACE = (argc > 2) ? atoi(argv[2]) : true;
  int INV = (argc > 3) ? atoi(argv[3]) : true;

  const ntt::Ordering ordering = ntt::Ordering::kNN;
  const char* ordering_str = ordering == ntt::Ordering::kNN   ? "NN"
                             : ordering == ntt::Ordering::kNR ? "NR"
                             : ordering == ntt::Ordering::kRN ? "RN"
                                                              : "RR";

  printf("running ntt 2^%d, INV=%d, ordering=%s, inplace=%d\n", NTT_LOG_SIZE, INV, ordering_str, INPLACE);

  hipFree(nullptr); // init GPU context (warmup)

  // init domain
  auto ntt_config = ntt::DefaultNTTConfig<test_scalar>();
  ntt_config.ordering = ordering;
  ntt_config.are_inputs_on_device = true;
  ntt_config.are_outputs_on_device = true;

  $CUDA(hipEventCreate(&icicle_start));
  $CUDA(hipEventCreate(&icicle_stop));
  $CUDA(hipEventCreate(&new_start));
  $CUDA(hipEventCreate(&new_stop));

  auto start = std::chrono::high_resolution_clock::now();
  const test_scalar basic_root = test_scalar::omega(NTT_LOG_SIZE);
  ntt::InitDomain(basic_root, ntt_config.ctx);
  auto stop = std::chrono::high_resolution_clock::now();
  auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
  std::cout << "initDomain took: " << duration / 1000 << " MS" << std::endl;

  // cpu allocation
  auto CpuScalars = std::make_unique<test_data[]>(NTT_SIZE);
  auto CpuOutputOld = std::make_unique<test_data[]>(NTT_SIZE);
  auto CpuOutputNew = std::make_unique<test_data[]>(NTT_SIZE);

  // gpu allocation
  test_data *GpuScalars, *GpuOutputOld, *GpuOutputNew;
  $CUDA(hipMalloc(&GpuScalars, sizeof(test_data) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputOld, sizeof(test_data) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputNew, sizeof(test_data) * NTT_SIZE));

  // init inputs
  incremental_values(CpuScalars.get(), NTT_SIZE);
  $CUDA(hipMemcpy(GpuScalars, CpuScalars.get(), NTT_SIZE, hipMemcpyHostToDevice));

  // inplace
  if (INPLACE) { $CUDA(hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice)); }

  // run ntt
  auto benchmark = [&](bool is_print, int iterations) {
    // NEW
    $CUDA(hipEventRecord(new_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = false; // mixed-radix ntt (a.k.a new ntt)
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(
        INPLACE ? GpuOutputNew : GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config,
        GpuOutputNew);
    }
    $CUDA(hipEventRecord(new_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&new_time, new_start, new_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    // OLD
    $CUDA(hipEventRecord(icicle_start, ntt_config.ctx.stream));
    ntt_config.is_force_radix2 = true;
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
    }
    $CUDA(hipEventRecord(icicle_stop, ntt_config.ctx.stream));
    $CUDA(hipStreamSynchronize(ntt_config.ctx.stream));
    $CUDA(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
    if (is_print) { fprintf(stderr, "cuda err %d\n", hipGetLastError()); }

    if (is_print) {
      printf("Old Runtime=%0.3f MS\n", icicle_time / iterations);
      printf("New Runtime=%0.3f MS\n", new_time / iterations);
    }
  };

  benchmark(false /*=print*/, 1); // warmup
  int count = INPLACE ? 1 : 1;
  if (INPLACE) { $CUDA(hipMemcpy(GpuOutputNew, GpuScalars, NTT_SIZE * sizeof(test_data), hipMemcpyDeviceToDevice)); }
  benchmark(true /*=print*/, count);

  // verify
  $CUDA(hipMemcpy(CpuOutputNew.get(), GpuOutputNew, NTT_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(CpuOutputOld.get(), GpuOutputOld, NTT_SIZE * sizeof(test_data), hipMemcpyDeviceToHost));

  bool success = true;
  for (int i = 0; i < NTT_SIZE; i++) {
    if (CpuOutputNew[i] != CpuOutputOld[i]) {
      success = false;
      std::cout << i << " ref " << CpuOutputOld[i] << " != " << CpuOutputNew[i] << std::endl;
      break;
    } else {
      // std::cout << i << " ref " << CpuOutputOld[i] << " == " << CpuOutputNew[i] << std::endl;
      // break;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  printf("%s\n", success_str);

  $CUDA(hipFree(GpuScalars));
  $CUDA(hipFree(GpuOutputOld));
  $CUDA(hipFree(GpuOutputNew));

  return 0;
}