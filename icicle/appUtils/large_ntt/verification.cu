#include "hip/hip_runtime.h"

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>
#include "../../curves/bls12_377/curve_config.cuh"
#include "ntt.cuh"
// #include "../../curves/bn254/curve_config.cuh"

// #include <stdio.h>
// #include <stdint.h>
// #include <hip/hip_cooperative_groups.h>

using namespace BLS12_377;
typedef scalar_t test_scalar;
#include "kernel_ntt.cu"

#define $CUDA(call) if((call)!=0) { printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError()); exit(1); }

void random_samples(test_scalar* res, uint32_t count) {
  for(int i=0;i<count;i++)
    res[i]= i<1000? test_scalar::rand_host() : res[i-1000];
    // res[i]= i<1000? test_scalar::omega(2) : res[i-1000];
}

void incremental_values(test_scalar* res, uint32_t count) {
  for(int i=0;i<count;i++)
    res[i]=i? res[i-1]+test_scalar::one() : test_scalar::zero();
}

int main(){

  int NTT_LOG_SIZE = 6;
  int TT_LOG_SIZE = 18;
  int NTT_SIZE = 1<<NTT_LOG_SIZE;
  int TT_SIZE = 1<<TT_LOG_SIZE;

  //cpu allocation
  test_scalar* cpuIcicle;
  uint4* cpuNew;
  cpuIcicle=(test_scalar*)malloc(sizeof(test_scalar)*NTT_SIZE);
  cpuNew=(uint4*)malloc(sizeof(uint4)*NTT_SIZE*2);
  if(cpuIcicle==NULL || cpuNew==NULL) {
    fprintf(stderr, "Malloc failed\n");
    exit(1);
  }

  //gpu allocation
  test_scalar* gpuIcicle;
  uint4* gpuNew;
  uint4* gpuTwiddles;
  $CUDA(hipMalloc((void**)&gpuIcicle, sizeof(test_scalar)*NTT_SIZE));
  $CUDA(hipMalloc((void**)&gpuNew, sizeof(uint4)*NTT_SIZE*2));
  $CUDA(hipMalloc((void**)&gpuTwiddles, sizeof(uint4)*TT_SIZE*2));

  //init inputs
  random_samples(cpuIcicle, NTT_SIZE);
  // incremental_values(cpuIcicle, NTT_SIZE);
  for (int i = 0; i < NTT_SIZE; i++)
  {
    cpuNew[i] = cpuIcicle[i].load_half(false);
    cpuNew[NTT_SIZE + i] = cpuIcicle[i].load_half(true);
  }
  $CUDA(hipMemcpy(gpuIcicle, cpuIcicle, sizeof(test_scalar)*NTT_SIZE, hipMemcpyHostToDevice));
  $CUDA(hipMemcpy(gpuNew, cpuNew, sizeof(uint4)*NTT_SIZE*2, hipMemcpyHostToDevice));
  generate_external_twiddles(gpuTwiddles, TT_LOG_SIZE);


  //run ntts
  ntt64<<<1, 8, 512*sizeof(uint4)>>>(gpuNew, gpuNew, NTT_LOG_SIZE ,1);
  ntt_end2end_batch_template<test_scalar, test_scalar>(gpuIcicle, NTT_SIZE, NTT_SIZE, false, 0);
  reverse_order_batch(gpuIcicle, NTT_SIZE, NTT_LOG_SIZE, 1, 0);
  
  //verify
  $CUDA(hipMemcpy(cpuIcicle, gpuIcicle, sizeof(test_scalar)*NTT_SIZE, hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(cpuNew, gpuNew, sizeof(uint4)*NTT_SIZE*2, hipMemcpyDeviceToHost));
  bool success = true;
  for (int i = 0; i < NTT_SIZE; i++)
  {
    test_scalar icicle_temp, new_temp;
    icicle_temp = cpuIcicle[i];
    new_temp.store_half(cpuNew[i], false);
    new_temp.store_half(cpuNew[i+NTT_SIZE], true);
    if (icicle_temp != new_temp){
      success = false;
      std::cout << "ref "<< icicle_temp << " != " << new_temp <<std::endl;
    }
    // else{
    //   std::cout << "ref "<< icicle_temp << " == " << new_temp <<std::endl;
    // }
  }
  if (success){
    printf("success!\n");
  }


  return 0;

}