
#define CURVE_ID 1 // TODO Yuval: move to makefile

#include "../../primitives/field.cuh"
#include "../../primitives/projective.cuh"
#include "../../utils/cuda_utils.cuh"
#include <chrono>
#include <iostream>
#include <vector>

#include "curves/curve_config.cuh"
#include "ntt/ntt.cu"
#include "large_ntt/large_ntt.cuh"
#include <memory>

#define PERFORMANCE

typedef curve_config::scalar_t test_scalar;
#include "kernel_ntt.cu"

#define $CUDA(call)                                                                                                    \
  if ((call) != 0) {                                                                                                   \
    printf("\nCall \"" #call "\" failed from %s, line %d, error=%d\n", __FILE__, __LINE__, hipGetLastError());        \
    exit(1);                                                                                                           \
  }

void random_samples(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i < 1000 ? test_scalar::rand_host() : res[i - 1000];
}

void incremental_values(test_scalar* res, uint32_t count)
{
  for (int i = 0; i < count; i++)
    res[i] = i ? res[i - 1] + test_scalar::one() * test_scalar::omega(4) : test_scalar::zero();
}

int main()
{
#ifdef PERFORMANCE
  hipEvent_t icicle_start, icicle_stop, new_start, new_stop;
  float icicle_time, new_time;
#endif

  int NTT_LOG_SIZE = 21;
  int NTT_SIZE = 1 << NTT_LOG_SIZE;
  int INV = false;
  const ntt::Ordering ordering = ntt::Ordering::kNN;
  const char* ordering_str = ordering == ntt::Ordering::kNN   ? "NN"
                             : ordering == ntt::Ordering::kNR ? "NR"
                             : ordering == ntt::Ordering::kRN ? "RN"
                                                              : "RR";

  printf("running ntt 2^%d, INV=%d, ordering=%s\n", NTT_LOG_SIZE, INV, ordering_str);

  // cpu allocation
  auto CpuScalars = std::make_unique<test_scalar[]>(NTT_SIZE);
  auto CpuOutputOld = std::make_unique<test_scalar[]>(NTT_SIZE);
  auto CpuOutputNew = std::make_unique<test_scalar[]>(NTT_SIZE);

  // gpu allocation
  test_scalar *GpuScalars, *GpuOutputOld, *GpuOutputNew;
  $CUDA(hipMalloc(&GpuScalars, sizeof(test_scalar) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputOld, sizeof(test_scalar) * NTT_SIZE));
  $CUDA(hipMalloc(&GpuOutputNew, sizeof(test_scalar) * NTT_SIZE));

  // init inputs
  random_samples(CpuScalars.get(), NTT_SIZE);
  $CUDA(hipMemcpy(GpuScalars, CpuScalars.get(), NTT_SIZE, hipMemcpyHostToDevice));

  // new algorithm init
  // ntt::MixedRadixNTT new_ntt(NTT_SIZE, INV, ordering);
  // old algorithm init
  auto ntt_config = ntt::DefaultNTTConfig<test_scalar>();
  ntt_config.ordering = ordering;
  ntt_config.are_inputs_on_device = true;
  ntt_config.are_outputs_on_device = true;
  ntt_config.is_force_radix2 = true; // to compare to radix2 algorithm
  const test_scalar basic_root = test_scalar::omega(NTT_LOG_SIZE);
  ntt::InitDomain(basic_root, ntt_config.ctx);

#ifdef PERFORMANCE
  $CUDA(hipEventCreate(&icicle_start));
  $CUDA(hipEventCreate(&icicle_stop));
  $CUDA(hipEventCreate(&new_start));
  $CUDA(hipEventCreate(&new_stop));

  // run ntt
  auto benchmark = [&](bool is_print, int iterations) {
    $CUDA(hipEventRecord(new_start, 0));
    for (size_t i = 0; i < iterations; i++) {
      // Note: measuring construction/destruction everytime since this is what real usecase is doing
      ntt::MixedRadixNTT new_ntt(NTT_SIZE, INV, ordering);
      new_ntt(GpuScalars, GpuOutputNew);
    }
    $CUDA(hipEventRecord(new_stop, 0));
    $CUDA(hipDeviceSynchronize());
    $CUDA(hipEventElapsedTime(&new_time, new_start, new_stop));
    hipDeviceSynchronize();
    if (is_print) { printf("cuda err %d\n", hipGetLastError()); }

    $CUDA(hipEventRecord(icicle_start, 0));
    for (size_t i = 0; i < iterations; i++) {
      ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
    }
    $CUDA(hipEventRecord(icicle_stop, 0));
    $CUDA(hipDeviceSynchronize());
    $CUDA(hipEventElapsedTime(&icicle_time, icicle_start, icicle_stop));
    hipDeviceSynchronize();
    if (is_print) {
      printf("cuda err %d\n", hipGetLastError());
      fprintf(stderr, "Old Runtime=%0.3f MS\n", icicle_time / iterations);
      fprintf(stderr, "New Runtime=%0.3f MS\n", new_time / iterations);
    }
  };

  int count = 1;
  benchmark(false /*=print*/, 1); // warmup - is this applicable to real usecase??
  benchmark(true /*=print*/, count);
#else
  new_ntt(GpuScalars, GpuOutputNew);
  hipDeviceSynchronize();
  printf("finished new\n");

  ntt::NTT(GpuScalars, NTT_SIZE, INV ? ntt::NTTDir::kInverse : ntt::NTTDir::kForward, ntt_config, GpuOutputOld);
  printf("finished old\n");

  // verify
  $CUDA(hipMemcpy(CpuOutputNew.get(), GpuOutputNew, NTT_SIZE, hipMemcpyDeviceToHost));
  $CUDA(hipMemcpy(CpuOutputOld.get(), GpuOutputOld, NTT_SIZE, hipMemcpyDeviceToHost));

  bool success = true;
  for (int i = 0; i < NTT_SIZE; i++) {
    if (CpuOutputNew[i] != CpuOutputOld[i]) {
      success = false;
      std::cout << i << " ref " << CpuOutputOld[i] << " != " << CpuOutputNew[i] << std::endl;
      // break;
    } else {
      // std::cout << i << " ref " << CpuOutputOld[i] << " == " << CpuOutputNew[i] << std::endl;
      // break;
    }
  }
  const char* success_str = success ? "SUCCESS!" : "FAIL!";
  printf("%s\n", success_str);
#endif

  $CUDA(hipFree(GpuScalars));
  $CUDA(hipFree(GpuOutputOld));
  $CUDA(hipFree(GpuOutputNew));

  return 0;
}