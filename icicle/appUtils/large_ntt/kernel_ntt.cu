#include "hip/hip_runtime.h"

#include "thread_ntt.cu"
#include "curves/curve_config.cuh"
#include "appUtils/large_ntt/large_ntt.cuh"

namespace ntt {

  __device__ uint32_t dig_rev(uint32_t num, uint32_t log_size, bool dit)
  {
    uint32_t rev_num = 0, temp, dig_len;
    if (dit) {
      for (int i = 4; i >= 0; i--) {
        dig_len = STAGE_SIZES_DEVICE[log_size][i];
        temp = num & ((1 << dig_len) - 1);
        num = num >> dig_len;
        rev_num = rev_num << dig_len;
        rev_num = rev_num | temp;
      }
    } else {
      for (int i = 0; i < 5; i++) {
        dig_len = STAGE_SIZES_DEVICE[log_size][i];
        temp = num & ((1 << dig_len) - 1);
        num = num >> dig_len;
        rev_num = rev_num << dig_len;
        rev_num = rev_num | temp;
      }
    }
    return rev_num;
  }

  __launch_bounds__(64) __global__
    void reorder_digits_kernel(uint4* arr, uint4* arr_reordered, uint32_t log_size, bool dit)
  {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    uint32_t rd = tid;
    uint32_t wr = dig_rev(tid, log_size, dit);
    arr_reordered[wr] = arr[rd];
    arr_reordered[wr + (1 << log_size)] = arr[rd + (1 << log_size)];
  }

  __launch_bounds__(64) __global__ void ntt64(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    uint32_t data_stride,
    uint32_t log_data_stride,
    uint32_t twiddle_stride,
    bool strided,
    uint32_t stage_num,
    bool inv,
    bool dit)
  {
    NTTEngine engine;
    stage_metadata s_meta;
    extern __shared__ uint4 shmem[];

    s_meta.th_stride = 8;
    s_meta.ntt_block_size = 64;
    s_meta.ntt_block_id = (blockIdx.x << 3) + (strided ? (threadIdx.x & 0x7) : (threadIdx.x >> 3));
    s_meta.ntt_inp_id = strided ? (threadIdx.x >> 3) : (threadIdx.x & 0x7);

    engine.loadBasicTwiddles(basic_twiddles);
    engine.loadGlobalData(in, data_stride, log_data_stride, log_size, strided, s_meta);
    if (twiddle_stride && dit) {
      engine.loadExternalTwiddles(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.loadInternalTwiddles(internal_twiddles, strided);

#pragma unroll 1
    for (uint32_t phase = 0; phase < 2; phase++) {
      engine.ntt8win();
      if (phase == 0) {
        engine.SharedData64Columns8(shmem, true, false, strided); // store low
        __syncthreads();
        engine.SharedData64Rows8(shmem, false, false, strided); // load low
        engine.SharedData64Rows8(shmem, true, true, strided);   // store high
        __syncthreads();
        engine.SharedData64Columns8(shmem, false, true, strided); // load high
        engine.twiddlesInternal();
      }
    }

    if (twiddle_stride && !dit) {
      engine.loadExternalTwiddles(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.storeGlobalData(out, data_stride, log_data_stride, log_size, strided, s_meta);
  }

  __launch_bounds__(64) __global__ void ntt32(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    uint32_t data_stride,
    uint32_t log_data_stride,
    uint32_t twiddle_stride,
    bool strided,
    uint32_t stage_num,
    bool inv,
    bool dit)
  {
    NTTEngine engine;
    stage_metadata s_meta;
    extern __shared__ uint4 shmem[];

    s_meta.th_stride = 4;
    s_meta.ntt_block_size = 32;
    s_meta.ntt_block_id = (blockIdx.x << 4) + (strided ? (threadIdx.x & 0xf) : (threadIdx.x >> 2));
    s_meta.ntt_inp_id = strided ? (threadIdx.x >> 4) : (threadIdx.x & 0x3);

    engine.loadBasicTwiddles(basic_twiddles);
    engine.loadGlobalData(in, data_stride, log_data_stride, log_size, strided, s_meta);
    engine.loadInternalTwiddles32(internal_twiddles, strided);
    engine.ntt8win();
    engine.twiddlesInternal();
    engine.SharedData32Columns8(shmem, true, false, strided); // store low
    __syncthreads();
    engine.SharedData32Rows4_2(shmem, false, false, strided); // load low
    engine.SharedData32Rows8(shmem, true, true, strided);     // store high
    __syncthreads();
    engine.SharedData32Columns4_2(shmem, false, true, strided); // load high
    engine.ntt4_2();
    if (twiddle_stride) {
      engine.loadExternalTwiddles32(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.storeGlobalData32(out, data_stride, log_data_stride, log_size, strided, s_meta);
  }

  __launch_bounds__(64) __global__ void ntt32dit(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    uint32_t data_stride,
    uint32_t log_data_stride,
    uint32_t twiddle_stride,
    bool strided,
    uint32_t stage_num,
    bool inv,
    bool dit)
  {
    NTTEngine engine;
    stage_metadata s_meta;
    extern __shared__ uint4 shmem[];

    s_meta.th_stride = 4;
    s_meta.ntt_block_size = 32;
    s_meta.ntt_block_id = (blockIdx.x << 4) + (strided ? (threadIdx.x & 0xf) : (threadIdx.x >> 2));
    s_meta.ntt_inp_id = strided ? (threadIdx.x >> 4) : (threadIdx.x & 0x3);

    engine.loadBasicTwiddles(basic_twiddles);
    engine.loadGlobalData32(in, data_stride, log_data_stride, log_size, strided, s_meta);
    if (twiddle_stride) {
      engine.loadExternalTwiddles32(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.loadInternalTwiddles32(internal_twiddles, strided);
    engine.ntt4_2();
    engine.SharedData32Columns4_2(shmem, true, false, strided); // store low
    __syncthreads();
    engine.SharedData32Rows8(shmem, false, false, strided); // load low
    engine.SharedData32Rows4_2(shmem, true, true, strided); // store high
    __syncthreads();
    engine.SharedData32Columns8(shmem, false, true, strided); // load high
    engine.twiddlesInternal();
    engine.ntt8win();
    engine.storeGlobalData(out, data_stride, log_data_stride, log_size, strided, s_meta);
  }

  __launch_bounds__(64) __global__ void ntt16(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    uint32_t data_stride,
    uint32_t log_data_stride,
    uint32_t twiddle_stride,
    bool strided,
    uint32_t stage_num,
    bool inv,
    bool dit)
  {
    NTTEngine engine;
    stage_metadata s_meta;
    extern __shared__ uint4 shmem[];

    s_meta.th_stride = 2;
    s_meta.ntt_block_size = 16;
    s_meta.ntt_block_id = (blockIdx.x << 5) + (strided ? (threadIdx.x & 0x1f) : (threadIdx.x >> 1));
    s_meta.ntt_inp_id = strided ? (threadIdx.x >> 5) : (threadIdx.x & 0x1);

    engine.loadBasicTwiddles(basic_twiddles);
    engine.loadGlobalData(in, data_stride, log_data_stride, log_size, strided, s_meta);
    engine.loadInternalTwiddles16(internal_twiddles, strided);
    engine.ntt8win();
    engine.twiddlesInternal();
    engine.SharedData16Columns8(shmem, true, false, strided); // store low
    __syncthreads();
    engine.SharedData16Rows2_4(shmem, false, false, strided); // load low
    engine.SharedData16Rows8(shmem, true, true, strided);     // store high
    __syncthreads();
    engine.SharedData16Columns2_4(shmem, false, true, strided); // load high
    engine.ntt2_4();
    if (twiddle_stride) {
      engine.loadExternalTwiddles16(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.storeGlobalData16(out, data_stride, log_data_stride, log_size, strided, s_meta);
  }

  __launch_bounds__(64) __global__ void ntt16dit(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    uint32_t data_stride,
    uint32_t log_data_stride,
    uint32_t twiddle_stride,
    bool strided,
    uint32_t stage_num,
    bool inv,
    bool dit)
  {
    NTTEngine engine;
    stage_metadata s_meta;
    extern __shared__ uint4 shmem[];

    s_meta.th_stride = 2;
    s_meta.ntt_block_size = 16;
    s_meta.ntt_block_id = (blockIdx.x << 5) + (strided ? (threadIdx.x & 0x1f) : (threadIdx.x >> 1));
    s_meta.ntt_inp_id = strided ? (threadIdx.x >> 5) : (threadIdx.x & 0x1);

    engine.loadBasicTwiddles(basic_twiddles);
    engine.loadGlobalData16(in, data_stride, log_data_stride, log_size, strided, s_meta);
    if (twiddle_stride) {
      engine.loadExternalTwiddles16(twiddles, twiddle_stride, strided, s_meta, log_size, stage_num);
      engine.twiddlesExternal();
    }
    engine.loadInternalTwiddles16(internal_twiddles, strided);
    engine.ntt2_4();
    engine.SharedData16Columns2_4(shmem, true, false, strided); // store low
    __syncthreads();
    engine.SharedData16Rows8(shmem, false, false, strided); // load low
    engine.SharedData16Rows2_4(shmem, true, true, strided); // store high
    __syncthreads();
    engine.SharedData16Columns8(shmem, false, true, strided); // load high
    engine.twiddlesInternal();
    engine.ntt8win();
    engine.storeGlobalData(out, data_stride, log_data_stride, log_size, strided, s_meta);
  }

  __global__ void normalize_kernel(uint4* data, uint32_t size, curve_config::scalar_t norm_factor)
  {
    curve_config::scalar_t temp;
    temp.store_half(data[threadIdx.x], false);
    temp.store_half(data[threadIdx.x + size], true);
    temp = temp * norm_factor;
    data[threadIdx.x] = temp.load_half(false);
    data[threadIdx.x + size] = temp.load_half(true);
  }

  __global__ void generate_base_table(curve_config::scalar_t basic_root, uint4* base_table, uint32_t skip)
  {
    curve_config::scalar_t w = basic_root;
    curve_config::scalar_t t = curve_config::scalar_t::one();
    for (int i = 0; i < 64; i += skip) {
      base_table[i] = t.load_half(false);
      base_table[i + 64] = t.load_half(true);
      t = t * w;
    }
  }

  __global__ void generate_basic_twiddles(curve_config::scalar_t basic_root, uint4* basic_twiddles)
  {
    curve_config::scalar_t w0 = basic_root * basic_root;
    curve_config::scalar_t w1 = (basic_root + w0 * basic_root) * curve_config::scalar_t::inv_log_size(1);
    curve_config::scalar_t w2 = (basic_root - w0 * basic_root) * curve_config::scalar_t::inv_log_size(1);
    basic_twiddles[0] = w0.load_half(false);
    basic_twiddles[3] = w0.load_half(true);
    basic_twiddles[1] = w1.load_half(false);
    basic_twiddles[4] = w1.load_half(true);
    basic_twiddles[2] = w2.load_half(false);
    basic_twiddles[5] = w2.load_half(true);
  }

  __global__ void generate_twiddle_combinations(
    uint4* w6_table,
    uint4* w12_table,
    uint4* w18_table,
    uint4* w24_table,
    uint4* w30_table,
    uint4* twiddles,
    uint32_t log_size,
    uint32_t stage_num,
    curve_config::scalar_t norm_factor)
  {
    uint32_t tid = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t range1 = 0, range2 = 0, ind;
    for (ind = 0; ind < stage_num; ind++)
      range1 += STAGE_SIZES_DEVICE[log_size][ind];
    range2 = STAGE_SIZES_DEVICE[log_size][ind];
    uint32_t root_order = range1 + range2;
    uint32_t exp = ((tid & ((1 << range1) - 1)) * (tid >> range1)) << (30 - root_order);
    curve_config::scalar_t w6, w12, w18, w24, w30;
    w6.store_half(w6_table[exp >> 24], false);
    w6.store_half(w6_table[(exp >> 24) + 64], true);
    w12.store_half(w12_table[((exp >> 18) & 0x3f)], false);
    w12.store_half(w12_table[((exp >> 18) & 0x3f) + 64], true);
    w18.store_half(w18_table[((exp >> 12) & 0x3f)], false);
    w18.store_half(w18_table[((exp >> 12) & 0x3f) + 64], true);
    w24.store_half(w24_table[((exp >> 6) & 0x3f)], false);
    w24.store_half(w24_table[((exp >> 6) & 0x3f) + 64], true);
    w30.store_half(w30_table[(exp & 0x3f)], false);
    w30.store_half(w30_table[(exp & 0x3f) + 64], true);
    curve_config::scalar_t t = w6 * w12 * w18 * w24 * w30 * norm_factor;
    twiddles[tid + LOW_W_OFFSETS[log_size][stage_num]] = t.load_half(false);
    twiddles[tid + HIGH_W_OFFSETS[log_size][stage_num]] = t.load_half(true);
  }

  void large_ntt(
    uint4* in,
    uint4* out,
    uint4* twiddles,
    uint4* internal_twiddles,
    uint4* basic_twiddles,
    uint32_t log_size,
    bool inv,
    bool dit)
  {
    // special cases:
    if (log_size == 1 || log_size == 2 || log_size == 3 || log_size == 7) {
      throw std::invalid_argument("size not implemented");
    }
    if (log_size == 4) {
      if (dit) {
        ntt16dit<<<1, 4, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      } else {
        ntt16<<<1, 4, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      }
      if (inv) normalize_kernel<<<1, 16>>>(out, 16, curve_config::scalar_t::inv_log_size(4));
      return;
    }
    if (log_size == 5) {
      if (dit) {
        ntt32dit<<<1, 4, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      } else {
        ntt32<<<1, 4, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      }
      if (inv) normalize_kernel<<<1, 32>>>(out, 32, curve_config::scalar_t::inv_log_size(5));
      return;
    }
    if (log_size == 6) {
      ntt64<<<1, 8, 8 * 64 * sizeof(uint4)>>>(
        in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      if (inv) normalize_kernel<<<1, 64>>>(out, 64, curve_config::scalar_t::inv_log_size(6));
      return;
    }
    if (log_size == 8) {
      if (dit)
        ntt16dit<<<1, 32, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      if (dit)
        ntt16dit<<<1, 64, 8 * 64 * sizeof(uint4)>>>(
          out, out, twiddles, internal_twiddles, basic_twiddles, log_size, 16, 4, 16, true, 1, inv,
          dit); // we need threads 32+ although 16-31 are idle
      if (!dit)
        ntt16<<<1, 64, 8 * 64 * sizeof(uint4)>>>(
          in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 16, 4, 16, true, 1, inv,
          dit); // we need threads 32+ although 16-31 are idle
      if (!dit)
        ntt16<<<1, 32, 8 * 64 * sizeof(uint4)>>>(
          out, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1, 0, 0, false, 0, inv, dit);
      return;
    }

    // general case:
    if (dit) {
      for (int i = 0; i < 5; i++) {
        uint32_t stage_size = STAGE_SIZES_HOST[log_size][i];
        uint32_t stride_log = 0;
        for (int j = 0; j < i; j++)
          stride_log += STAGE_SIZES_HOST[log_size][j];
        if (stage_size == 6)
          ntt64<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            i ? out : in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log, stride_log,
            i ? (1 << stride_log) : 0, i, i, inv, dit);
        if (stage_size == 5)
          ntt32dit<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            i ? out : in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log, stride_log,
            i ? (1 << stride_log) : 0, i, i, inv, dit);
        if (stage_size == 4)
          ntt16dit<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            i ? out : in, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log, stride_log,
            i ? (1 << stride_log) : 0, i, i, inv, dit);
      }
    } else {
      bool first_run = false, prev_stage = false;
      for (int i = 4; i >= 0; i--) {
        uint32_t stage_size = STAGE_SIZES_HOST[log_size][i];
        uint32_t stride_log = 0;
        for (int j = 0; j < i; j++)
          stride_log += STAGE_SIZES_HOST[log_size][j];
        first_run = stage_size && !prev_stage;
        if (stage_size == 6)
          ntt64<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            first_run ? in : out, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log,
            stride_log, i ? (1 << stride_log) : 0, i, i, inv, dit);
        if (stage_size == 5)
          ntt32<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            first_run ? in : out, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log,
            stride_log, i ? (1 << stride_log) : 0, i, i, inv, dit);
        if (stage_size == 4)
          ntt16<<<1 << (log_size - 9), 64, 8 * 64 * sizeof(uint4)>>>(
            first_run ? in : out, out, twiddles, internal_twiddles, basic_twiddles, log_size, 1 << stride_log,
            stride_log, i ? (1 << stride_log) : 0, i, i, inv, dit);
        prev_stage = stage_size;
      }
    }
  }

  /*================================ MixedRadixNTT =========================================*/
  MixedRadixNTT::MixedRadixNTT(int ntt_size, bool is_inverse, Ordering ordering, hipStream_t hip_stream)
      : m_ntt_size(ntt_size), m_ntt_log_size(int(log2(ntt_size))), m_is_inverse(is_inverse), m_ordering(ordering),
        m_cuda_stream(hip_stream)
  {
    hipError_t err_result = init();
    if (err_result != hipSuccess) throw(IcicleError(err_result, "CUDA error"));
  }

  hipError_t MixedRadixNTT::init()
  {
    CHK_IF_RETURN(
      hipMalloc(&m_gpuTwiddles, sizeof(uint4) * (m_ntt_size + 2 * (m_ntt_size >> 4)) * 2)); // TODO - sketchy
    CHK_IF_RETURN(hipMalloc(&m_gpuBasicTwiddles, sizeof(uint4) * 3 * 2));

    const auto basic_root =
      m_is_inverse ? curve_config::scalar_t::omega_inv(m_ntt_log_size) : curve_config::scalar_t::omega(m_ntt_log_size);
    CHK_IF_RETURN(generate_external_twiddles(basic_root));

    // temp memory for algorithm
    CHK_IF_RETURN(hipMalloc(&m_gpuMemA, sizeof(uint4) * m_ntt_size * 2));
    CHK_IF_RETURN(hipMalloc(&m_gpuMemB, sizeof(uint4) * m_ntt_size * 2));

    return CHK_LAST();
  }

  hipError_t MixedRadixNTT::generate_external_twiddles(curve_config::scalar_t basic_root)
  {
    CHK_IF_RETURN(hipMalloc(&m_w6_table, sizeof(uint4) * 64 * 2));
    CHK_IF_RETURN(hipMalloc(&m_w12_table, sizeof(uint4) * 64 * 2));
    CHK_IF_RETURN(hipMalloc(&m_w18_table, sizeof(uint4) * 64 * 2));
    CHK_IF_RETURN(hipMalloc(&m_w24_table, sizeof(uint4) * 64 * 2));
    CHK_IF_RETURN(hipMalloc(&m_w30_table, sizeof(uint4) * 64 * 2));

    curve_config::scalar_t temp_root = basic_root;
    generate_base_table<<<1, 1>>>(basic_root, m_w30_table, 1 << (30 - m_ntt_log_size));
    if (m_ntt_log_size > 24)
      for (int i = 0; i < 6 - (30 - m_ntt_log_size); i++)
        temp_root = temp_root * temp_root;
    generate_base_table<<<1, 1>>>(temp_root, m_w24_table, 1 << (m_ntt_log_size > 24 ? 0 : 24 - m_ntt_log_size));
    if (m_ntt_log_size > 18)
      for (int i = 0; i < 6 - (m_ntt_log_size > 24 ? 0 : 24 - m_ntt_log_size); i++)
        temp_root = temp_root * temp_root;
    generate_base_table<<<1, 1>>>(temp_root, m_w18_table, 1 << (m_ntt_log_size > 18 ? 0 : 18 - m_ntt_log_size));
    if (m_ntt_log_size > 12)
      for (int i = 0; i < 6 - (m_ntt_log_size > 18 ? 0 : 18 - m_ntt_log_size); i++)
        temp_root = temp_root * temp_root;
    generate_base_table<<<1, 1>>>(temp_root, m_w12_table, 1 << (m_ntt_log_size > 12 ? 0 : 12 - m_ntt_log_size));
    if (m_ntt_log_size > 6)
      for (int i = 0; i < 6 - (m_ntt_log_size > 12 ? 0 : 12 - m_ntt_log_size); i++)
        temp_root = temp_root * temp_root;
    generate_base_table<<<1, 1>>>(temp_root, m_w6_table, 1 << (m_ntt_log_size > 6 ? 0 : 6 - m_ntt_log_size));
    for (int i = 0; i < 3 - (m_ntt_log_size > 6 ? 0 : 6 - m_ntt_log_size); i++)
      temp_root = temp_root * temp_root;
    generate_basic_twiddles<<<1, 1>>>(temp_root, m_gpuBasicTwiddles);

    uint32_t temp = STAGE_SIZES_HOST[m_ntt_log_size][0];
    for (int i = 1; i < 5; i++) {
      if (!STAGE_SIZES_HOST[m_ntt_log_size][i]) break;
      temp += STAGE_SIZES_HOST[m_ntt_log_size][i];
      generate_twiddle_combinations<<<1 << (temp - 8), 256>>>(
        m_w6_table, m_w12_table, m_w18_table, m_w24_table, m_w30_table, m_gpuTwiddles, m_ntt_log_size, i,
        (temp == m_ntt_log_size && m_is_inverse) ? curve_config::scalar_t::inv_log_size(m_ntt_log_size)
                                                 : curve_config::scalar_t::one());
    }
    m_gpuIntTwiddles = m_w6_table;

    return CHK_LAST();
  }

  MixedRadixNTT::~MixedRadixNTT()
  {
    hipFreeAsync(m_gpuTwiddles, m_cuda_stream);
    hipFreeAsync(m_gpuBasicTwiddles, m_cuda_stream);
    hipFreeAsync(m_w6_table, m_cuda_stream);
    hipFreeAsync(m_w12_table, m_cuda_stream);
    hipFreeAsync(m_w18_table, m_cuda_stream);
    hipFreeAsync(m_w24_table, m_cuda_stream);
    hipFreeAsync(m_w30_table, m_cuda_stream);
    hipFreeAsync(m_gpuMemA, m_cuda_stream);
    hipFreeAsync(m_gpuMemB, m_cuda_stream);
  }

  template <typename E>
  static __global__ void copy_input_large_ntt(E* input, uint4* gpuMemA, int ntt_size)
  {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= ntt_size) return;
    gpuMemA[tid] = input[tid].load_half(false);
    gpuMemA[ntt_size + tid] = input[tid].load_half(true);
  }

  template <typename E>
  static __global__ void copy_output_large_ntt(uint4* gpuMemA, E* output, int ntt_size)
  {
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= ntt_size) return;
    output[tid].store_half(gpuMemA[tid], false);
    output[tid].store_half(gpuMemA[ntt_size + tid], true);
  }

  template <typename E>
  hipError_t MixedRadixNTT::operator()(E* d_input, E* d_output) // TODO fix names
  {
    CHK_INIT_IF_RETURN();

    const int NOF_BLOCKS = (1 << (max(m_ntt_log_size, 6) - 6));
    const int NOF_THREADS = min(64, 1 << m_ntt_log_size);

    copy_input_large_ntt<<<NOF_BLOCKS, NOF_THREADS>>>(d_input, m_gpuMemA, m_ntt_size);

    const bool reverse_input = m_ordering == Ordering::kNN;
    const bool reverse_output = m_ordering == Ordering::kRR;
    const bool is_dit = m_ordering == Ordering::kNN || m_ordering == Ordering::kRN;

    if (reverse_input) {
      reorder_digits_kernel<<<NOF_BLOCKS, NOF_THREADS>>>(m_gpuMemA, m_gpuMemB, m_ntt_log_size, is_dit);
    }

    uint4* ntt_input = reverse_input ? m_gpuMemB : m_gpuMemA;
    uint4* ntt_output = reverse_input ? m_gpuMemA : m_gpuMemB;
    large_ntt(
      ntt_input, ntt_output, m_gpuTwiddles, m_gpuIntTwiddles, m_gpuBasicTwiddles, m_ntt_log_size, m_is_inverse, is_dit);

    if (reverse_output) {
      reorder_digits_kernel<<<NOF_BLOCKS, NOF_THREADS>>>(ntt_output, ntt_input, m_ntt_log_size, is_dit);
      ntt_output = ntt_input;
    }

    copy_output_large_ntt<<<NOF_BLOCKS, NOF_THREADS>>>(ntt_output, d_output, m_ntt_size);

    return CHK_LAST();
  }

  // Explicit instantiation for scalar type
  template hipError_t
  MixedRadixNTT::operator()<curve_config::scalar_t>(curve_config::scalar_t*, curve_config::scalar_t*);

} // namespace ntt
