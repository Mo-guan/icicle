#include "hip/hip_runtime.h"
/*

Copyright (c) 2023 Yrrid Software, Inc.

Permission is hereby granted, free of charge, to any person obtaining a copy of this software and associated
documentation files (the �Software�), to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies of the Software,
and to permit persons to whom the Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all copies or substantial portions
of the Software.

THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE AUTHORS OR
COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.

*/

// #include <stdio.h>
// #include <stdint.h>
// #include "asm.cu"
// #include "Sampled96.cu"

// typedef Sampled96 Math;

#include "thread_ntt.cu"

// EXCHANGE_OFFSET = 129*64*8

#define DATA_OFFSET 0

__launch_bounds__(384)
__global__ void ntt1024(uint64_t* out, uint64_t* in, uint32_t* next, uint32_t count) {
  NTTEngine32 engine;
  uint32_t    dataIndex=0;

  #ifdef COMPUTE_ONLY
    bool        first=true;
  #endif
  
  engine.initializeRoot();
    
  while(true) {
    if((threadIdx.x & 0x1F)==0)
      dataIndex=atomicAdd(next, 1);
    dataIndex=__shfl_sync(0xFFFFFFFF, dataIndex, 0);  //send value to all threads in warp    
    if(dataIndex<count) {
      #if defined(COMPUTE_ONLY)
        if(first)
          engine.loadGlobalData(in, dataIndex);
        first=false;
      #else
        engine.loadGlobalData(in, dataIndex);
      #endif
    }
    else {
      if(dataIndex==count + (gridDim.x*blockDim.x>>5) - 1) { //didn't understand condition
        // last one to finish, reset the counter
        atomicExch(next, 0);
      }
      return;
    }
    #pragma unroll 1
    for(uint32_t phase=0;phase<2;phase++) {
      // ntt32 produces a lot of instructions, so we put this in a loop
      engine.ntt32(); 
      if(phase==0) {
        engine.storeSharedData(DATA_OFFSET);
        __syncwarp(0xFFFFFFFF);
        engine.loadSharedDataAndTwiddle32x32(DATA_OFFSET);
      }
    }
    engine.storeGlobalData(out, dataIndex);
  }
}


__global__ void thread_ntt_kernel(test_scalar* out, test_scalar* in, uint32_t* next, uint32_t count) {
  NTTEngine engine;
  uint32_t    dataIndex=blockIdx.x*blockDim.x+threadIdx.x;

  #ifdef COMPUTE_ONLY
    bool        first=true;
  #endif
  
  // engine.initializeRoot();
  engine.loadGlobalDataDep(in, dataIndex);
  // engine.ntt4_4();
  // for (int i = 0; i < 16; i++)
  // {
  //   engine.X[i] = in[i];
  // }
  

  // for (int i = 0; i < 100; i++)
  // {
    // engine.ntt16win_lowreg();
    // engine.ntt8_2();
    // engine.ntt8_2();
    // engine.ntt16();
    // engine.ntt16_win8ct2();
    engine.ntt16win();
    // engine.X[2] = engine.X[2]*engine.X[0];
  // }
  // engine.ntt16();
  // engine.X[0] = engine.X[0] + engine.X[1];
  // out[0] = out[0] + out[1];
  // out[0] = test_scalar::zero();
  // engine.storeGlobalData(out, dataIndex);
  // engine.storeGlobalData8_2(out, dataIndex);
  // engine.storeGlobalData16(out, dataIndex);
    
  // while(true) {
  //   if((threadIdx.x & 0x1F)==0)
  //     dataIndex=atomicAdd(next, 1);
  //   dataIndex=__shfl_sync(0xFFFFFFFF, dataIndex, 0);      
  //   if(dataIndex<count) {
  //     #if defined(COMPUTE_ONLY)
  //       if(first)
  //         engine.loadGlobalData(in, dataIndex);
  //       first=false;
  //     #else
  //       engine.loadGlobalData(in, dataIndex);
  //     #endif
  //   }
  //   else {
  //     if(dataIndex==count + (gridDim.x*blockDim.x>>5) - 1) {
  //       // last one to finish, reset the counter
  //       atomicExch(next, 0);
  //     }
  //     return;
  //   }
  //   #pragma unroll 1
  //   for(uint32_t phase=0;phase<2;phase++) {
  //     // ntt32 produces a lot of instructions, so we put this in a loop
  //     engine.ntt32(); 
  //     if(phase==0) {
  //       engine.storeSharedData(DATA_OFFSET);
  //       __syncwarp(0xFFFFFFFF);
  //       engine.loadSharedDataAndTwiddle32x32(DATA_OFFSET);
  //     }
  //   }
    engine.storeGlobalDataDep(out, dataIndex);
  // }
}

__launch_bounds__(64)
// __global__ void ntt_kernel_split_transpose(test_scalar* out, test_scalar* in) {
__global__ void ntt_kernel_split_transpose(uint4* out, uint4* in) {
  NTTEngine engine;
  uint32_t    dataIndex=blockIdx.x*blockDim.x+threadIdx.x;

  // if (blockIdx.x !=1) return;

  // if (blockIdx.x ==0 && threadIdx.x ==0) printf("start kernel\n");  
  // __shared__ uint4 shmem[2048*3];
  extern __shared__ uint4 shmem[];
  // if (blockIdx.x ==0 && threadIdx.x ==0) printf("shmem\n");

  // #ifdef COMPUTE_ONLY
  //   bool        first=true;
  // #endif
  
  // engine.initializeRoot();
  // engine.loadGlobalData(in, dataIndex);
  // engine.ntt4_4();
  // for (int i = 0; i < 100000; i++)
  // {
    // engine.ntt16win();
    // engine.ntt16win_lowreg();
    // engine.ntt8_2();
    // engine.ntt8_2();
    // engine.ntt16();
  // engine.ntt16_win8ct2();
  // }
  // engine.ntt16();
  // engine.X[0] = engine.X[0] + engine.X[1];
  // out[0] = out[0] + out[1];
  // out[0] = test_scalar::zero();
  // engine.storeGlobalData(out, dataIndex);
  // engine.storeGlobalData8_2(out, dataIndex);
  // engine.storeGlobalData16(out, dataIndex);
    
  // while(true) {
    // if((threadIdx.x & 0x1F)==0)
    //   dataIndex=atomicAdd(next, 1);
    // dataIndex=__shfl_sync(0xFFFFFFFF, dataIndex, 0);      
    // if(dataIndex<count) {
    //   #if defined(COMPUTE_ONLY)
    //     if(first)
    //       engine.loadGlobalData(in, dataIndex);
    //     first=false;
    //   #else
    //     engine.loadGlobalData(in, dataIndex);
    //   #endif
    // }
    // else {
    //   if(dataIndex==count + (gridDim.x*blockDim.x>>5) - 1) {
    //     // last one to finish, reset the counter
    //     atomicExch(next, 0);
    //   }
    //   return;
    // }
    // if (threadIdx.x!=0) return;
    // engine.loadGlobalDataDep(in, dataIndex);
    engine.loadGlobalData(in,blockIdx.x*512*2,1,64*8); //todo - change function to fit global ntt
    // engine.loadGlobalData(in,blockIdx.x*512*2,1,256*8); //todo - change function to fit global ntt
    // __syncthreads();
    // if (blockIdx.x ==0 && threadIdx.x ==0) printf("load global\n");
    // engine.externalTwiddles(); //todo
    // engine.twiddles256();
    // engine.ntt16_win8ct2();
    // engine.twiddles256();
    // engine.ntt16_win8ct2();
    // #pragma unroll 1
    // for (uint32_t i=0;i<100;i++) {
    #pragma unroll 1
    for (uint32_t phase=0;phase<2;phase++) {
      // ntt32 produces a lot of instructions, so we put this in a loop
      // engine.ntt16_win8ct2();
      // engine.plus();
      // engine.twiddles256();
      // engine.load_twiddles(threadIdx.x&0x7);
      // engine.twiddles64(threadIdx.x&0x7);
      // engine.ntt16_win8ct2();
      // engine.ntt8win();
      // engine.ntt16_win8ct2();
      // engine.twiddles256();
      // engine.ntt16_win8ct2();
      // engine.ntt16win();
      // engine.twiddles256();
      // engine.ntt16();
      // if(phase==0) {
      //   engine.SharedDataColumns2(shmem, true, false); //store low
      //   __syncthreads();
      //   // if (blockIdx.x ==0 && threadIdx.x ==0) printf("store shmem low\n");
      //   // if (blockIdx.x ==0 && threadIdx.x ==0){
      //   //   for (int i = 0; i < 512; i++)
      //   //   {
      //   //     if (i%32==0) printf("\n");
      //   //     if (i%256==0) printf("\n");
      //   //     printf("%d, ",shmem[i].w);
      //   //   }
      //   // }
      //   // __syncthreads();
      //   engine.SharedDataRows2(shmem, false, false); //load low
      //   // if (blockIdx.x ==0 && threadIdx.x ==0) printf("load shmem low\n");
      //   // __syncthreads(); //can avoid with switching rows and columns
      //   // if (blockIdx.x ==0 && threadIdx.x ==1){
      //   //   for (int i = 0; i < 16; i++)
      //   //   {
      //   //     printf("\n");
      //   //     printf("%d, ",engine.X[i].limbs_storage.limbs[0]);
      //   //   }
      //   // }
      //   engine.SharedDataRows2(shmem, true, true); //store high
      //   __syncthreads();
      //   // if (blockIdx.x ==0 && threadIdx.x ==0) printf("store shmem high\n");
      //   // if (blockIdx.x ==0 && threadIdx.x ==0){
      //   //   for (int i = 0; i < 2048; i++)
      //   //   {
      //   //     if (i%16==0) printf("\n");
      //   //     if (i%256==0) printf("\n");
      //   //     printf("%d, ",shmem[i].w);
      //   //   }
      //   // }
      //   // __syncthreads();
      //   engine.SharedDataColumns2(shmem, false, true); //load high
      //   // if (blockIdx.x ==0 && threadIdx.x ==0) printf("load shmem high\n");
      //   // engine.twiddles256();
      //   // engine.ntt16_win8ct2();
      //   // engine.twiddles256();
      // }
    // }
    }
    // #pragma unroll 1
    // for (uint32_t i=0;i<100*2;i++) {
    // // #pragma unroll 1
    // // for (uint32_t phase=0;phase<2;phase++) {
    //   // ntt32 produces a lot of instructions, so we put this in a loop
    //   // engine.ntt16_win8ct2();
    //   // if (i%2) engine.twiddles256();
    //   engine.twiddles256();
    //   engine.ntt16_win8ct2();
    //   // engine.ntt16win();
    // }
    engine.storeGlobalData(out,blockIdx.x*512*2,1,64*8); //todo - change function to fit global ntt
    // engine.storeGlobalData(out,blockIdx.x*512*2,1,256*8); //todo - change function to fit global ntt
    // engine.storeGlobalDataDep(out, dataIndex); //todo - change function to fit global ntt
  // }
}

__launch_bounds__(64)
__global__ void ntt64(uint4* out, uint4* in, uint32_t size, uint32_t stride) {
  NTTEngine engine;
  extern __shared__ uint4 shmem[];
  
  engine.initializeRoot(stride>1);
    
  #pragma unroll 1
  for (int i = 0; i < 1; i++) //todo - function of size
  {
    engine.loadGlobalData(in,blockIdx.x*64*8,stride,size); //todo - parametize

    #pragma unroll 1
    for(uint32_t phase=0;phase<2;phase++) {
      // this code produces a lot of instructions, so we put this in a loop
      // engine.twiddles64();
      engine.ntt8win(); 
      if(phase==0) {
        engine.SharedDataColumns2(shmem, true, false); //store low
        __syncthreads();
        engine.SharedDataRows2(shmem, false, false); //load low
        engine.SharedDataRows2(shmem, true, true); //store high
        __syncthreads();
        engine.SharedDataColumns2(shmem, false, true); //load high
        engine.twiddles64();
      }
    }

    engine.storeGlobalData(in,blockIdx.x*64*8,stride,size);
  }
}