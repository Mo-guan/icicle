#include "hip/hip_runtime.h"
#ifndef MSM
#define MSM
#pragma once
#include <stdexcept>
#include <hip/hip_runtime.h>
#include "../../primitives/affine.cuh"
#include <iostream>
#include <vector>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/device/device_scan.cuh>
#include "../../utils/cuda_utils.cuh"
#include "../../primitives/projective.cuh"
#include "../../primitives/field.cuh"
#include "msm.cuh"


#define BIG_TRIANGLE
// #define SSM_SUM  //WIP

template <typename T>
static constexpr __device__ __forceinline__ T ld_single(const T *ptr) {
return __ldg(ptr);
};

template <class T, typename U, unsigned STRIDE>
static constexpr __device__ __forceinline__ T ld(const T *address, const unsigned offset) {
  static_assert(alignof(T) % alignof(U) == 0);
  static_assert(sizeof(T) % sizeof(U) == 0);
  constexpr size_t count = sizeof(T) / sizeof(U);
  T result = {};
  auto pa = reinterpret_cast<const U *>(address) + offset;
  auto pr = reinterpret_cast<U *>(&result);
#pragma unroll
  for (unsigned i = 0; i < count; i++) {
    const auto pai = pa + i * STRIDE;
    const auto pri = pr + i;
    *pri = ld_single<U>(pai);
  }
  return result;
}

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<sizeof(T) % sizeof(uint4) == 0, uint4>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] uint4 _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<(sizeof(T) % sizeof(uint4) != 0) && (sizeof(T) % sizeof(uint2) == 0), uint2>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] uint2 _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

template <class T, unsigned STRIDE = 1, typename U = std::enable_if_t<sizeof(T) % sizeof(uint2) != 0, unsigned>>
static constexpr __device__ __forceinline__ T memory_load(const T *address, const unsigned offset = 0, [[maybe_unused]] unsigned _dummy = {}) {
  return ld<T, U, STRIDE>(address, offset);
};

//this kernel performs single scalar multiplication
//each thread multilies a single scalar and point
template <typename P, typename S>
__global__ void ssm_kernel(S *scalars, P *points, P *results, unsigned N) {

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) results[tid] = scalars[tid]*points[tid];

}

//this kernel sums all the elements in a given vector using multiple threads
template <typename P>
__global__ void sum_reduction_kernel(P *v, P* v_r) {

	unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;

	// Start at 1/2 block stride and divide by two each iteration
	for (unsigned s = blockDim.x / 2; s > 0; s >>= 1) {
		// Each thread does work unless it is further than the stride
		if (threadIdx.x < s) {
			v[tid] = v[tid] + v[tid + s];
		}
    __syncthreads();
	}

	// Let the thread 0 for this block write the final result
	if (threadIdx.x == 0) {
		v_r[blockIdx.x] = v[tid];
	}
}

//this kernel initializes the buckets with zero points
//each thread initializes a different bucket
template <typename P>
__global__ void initialize_buckets_kernel(P *buckets, unsigned N) {
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) buckets[tid] = P::zero(); //zero point

}

//this kernel splits the scalars into digits of size c
//each thread splits a single scalar into nof_bms digits
template <typename S>
__global__ void split_scalars_kernel(unsigned *buckets_indices, unsigned *point_indices, S *scalars, unsigned total_size, unsigned msm_log_size, unsigned nof_bms, unsigned bm_bitsize, unsigned c, unsigned top_bm_nof_missing_bits){
  
  constexpr unsigned sign_mask = 0x80000000;
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  unsigned bucket_index;
  unsigned bucket_index2;
  unsigned current_index;
  unsigned msm_index = tid >> msm_log_size;
  unsigned borrow = 0;
  if (tid < total_size){
    S scalar = scalars[tid];
    if (tid == 0) printf("scalar %u", scalar);

    for (unsigned bm = 0; bm < nof_bms; bm++)
    {
      // bucket_index = scalar.get_scalar_digit(bm, c) + (bm==nof_bms-1? ((tid&top_bm_nof_missing_bits)<<(c-top_bm_nof_missing_bits)) : 0);
      bucket_index = scalar.get_scalar_digit(bm, c);
      bucket_index += borrow;
      borrow = 0;
      unsigned sign = 0;
      if (tid == 0) printf("index %u", bucket_index);
      if (bucket_index > (1<<(c-1))) {
        bucket_index = (1 << c) - bucket_index;
        borrow = 1;
        sign = sign_mask;
      }
      if (tid == 0) printf("new index %u", bucket_index);
      // if (bm==nof_bms-1) {
      //   bucket_index2 = bucket_index + ((tid&((1<<top_bm_nof_missing_bits)-1))<<(c-top_bm_nof_missing_bits));
      //   if (tid<10) printf("tid %u bi1 %u bi2 %u\n",tid, bucket_index, bucket_index2);
      //   bucket_index = bucket_index2;
      // }
      current_index = bm * total_size + tid;
      buckets_indices[current_index] = (msm_index<<(c+bm_bitsize)) | (bm<<c) | bucket_index;  //the bucket module number and the msm number are appended at the msbs
      // buckets_indices[current_index] = (msm_index<<(c-1+bm_bitsize)) | (bm<<(c-1)) | bucket_index;  //the bucket module number and the msm number are appended at the msbs
      // point_indices[current_index] = tid; //the point index is saved for later
      point_indices[current_index] = sign | tid; //the point index is saved for later
    }
  }
}

//this kernel adds up the points in each bucket
// __global__ void accumulate_buckets_kernel(P *__restrict__ buckets, unsigned *__restrict__ bucket_offsets,
  //  unsigned *__restrict__ bucket_sizes, unsigned *__restrict__ single_bucket_indices, unsigned *__restrict__ point_indices, A *__restrict__ points, unsigned nof_buckets, unsigned batch_size, unsigned msm_idx_shift){
template <typename P, typename A>
<<<<<<< HEAD
// __global__ void accumulate_buckets_kernel(P *__restrict__ buckets, unsigned *__restrict__ bucket_offsets,
              //  unsigned *__restrict__ bucket_sizes, unsigned *__restrict__ single_bucket_indices, unsigned *__restrict__ point_indices, A *__restrict__ points, unsigned nof_buckets, unsigned batch_size, unsigned msm_idx_shift){
__global__ void accumulate_buckets_kernel(P *buckets, unsigned *bucket_offsets, unsigned *bucket_sizes, unsigned *single_bucket_indices, unsigned *point_indices, A *points, unsigned nof_buckets, unsigned *nof_buckets_to_compute, unsigned msm_idx_shift){
=======
__global__ void accumulate_buckets_kernel(P *__restrict__ buckets, const unsigned *__restrict__ bucket_offsets, const unsigned *__restrict__ bucket_sizes, const unsigned *__restrict__ single_bucket_indices, const unsigned *__restrict__ point_indices, A *__restrict__ points, const unsigned nof_buckets, const unsigned *nof_buckets_to_compute, const unsigned msm_idx_shift, const unsigned c){
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  
  constexpr unsigned sign_mask = 0x80000000;
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
<<<<<<< HEAD
  if (tid >= *nof_buckets_to_compute){ 
    return;
  }
  unsigned msm_index = single_bucket_indices[tid]>>msm_idx_shift;
  unsigned bucket_index = msm_index * nof_buckets + (single_bucket_indices[tid]&((1<<msm_idx_shift)-1));
  unsigned bucket_offset = bucket_offsets[tid];
=======
  // if (tid>=*nof_buckets_to_compute || tid<11){ 
  if (tid>=*nof_buckets_to_compute){ 
    return;
  }
  const unsigned msm_index = single_bucket_indices[tid]>>msm_idx_shift;
  const unsigned bm_index = (single_bucket_indices[tid]&((1<<msm_idx_shift)-1))>>c;
  const unsigned bucket_index = msm_index * nof_buckets + bm_index * ((1<<(c-1))+1) + (single_bucket_indices[tid]&((1<<c)-1));
  const unsigned bucket_offset = bucket_offsets[tid];
  // if (tid<10) printf("tid %u size %u\n", tid, bucket_sizes[tid]);
  // if (tid==0) return;
  // if ((bucket_index>>20)==13) return;
  // if (bucket_sizes[tid]==16777216) printf("tid %u size %u bucket %u offset %u\n", tid, bucket_sizes[tid], bucket_index, bucket_offset);
  // const unsigned *indexes = point_indices + bucket_offset;
  P bucket = P::zero(); //todo: get rid of init buckets? no.. because what about buckets with no points
  // unsigned point_ind;
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  for (unsigned i = 0; i < bucket_sizes[tid]; i++)  //add the relevant points starting from the relevant offset up to the bucket size
  {
    // unsigned point_ind = *indexes++;
    // auto point = memory_load<A>(points + point_ind);
    // point_ind = point_indices[bucket_offset+i];
    // bucket = bucket + P::one();
    unsigned point_ind = point_indices[bucket_offset+i];
    unsigned sign = point_ind & sign_mask;
    point_ind &= ~sign_mask;
    // printf("tid %u sign %u point ind %u \n", tid,sign, point_ind);
    A point = points[point_ind];
    if (sign) point = A::neg(point);
    bucket = bucket + point;
    // const unsigned* pa = reinterpret_cast<const unsigned*>(points[point_ind]);
    // P point;
    // Dummy_Scalar scal;
    // scal.x = __ldg(pa);
    // point.x = scal;
    // bucket = bucket + point;
  }
  // buckets[tid] = bucket;
  buckets[bucket_index] = bucket;
}

//this kernel sums the entire bucket module
//each thread deals with a single bucket module
template <typename P>
__global__ void big_triangle_sum_kernel(P* buckets, P* final_sums, unsigned nof_bms, unsigned c){

  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
<<<<<<< HEAD
  if (tid >= nof_bms) return;
  P line_sum = buckets[(tid+1)*(1<<c)-1];
=======
  if (tid>=nof_bms) return;
  P line_sum = buckets[(tid+1)*((1<<c)+1)-1];
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  final_sums[tid] = line_sum;
  for (unsigned i = (1<<c)-1; i >0; i--)
  {
    line_sum = line_sum + buckets[tid*((1<<c)+1) + i];  //using the running sum method
    final_sums[tid] = final_sums[tid] + line_sum;
  }
}

//this kernel uses single scalar multiplication to multiply each bucket by its index
//each thread deals with a single bucket
template <typename P, typename S>
__global__ void ssm_buckets_kernel(P* buckets, unsigned* single_bucket_indices, unsigned nof_buckets, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_buckets) return;
  unsigned bucket_index = single_bucket_indices[tid];
  S scalar_bucket_multiplier;
  scalar_bucket_multiplier = {bucket_index&((1<<c)-1), 0, 0, 0, 0, 0, 0, 0}; //the index without the bucket module index
  buckets[bucket_index] = scalar_bucket_multiplier*buckets[bucket_index];

}

//this kernel computes the final result using the double and add algorithm
//it is done by a single thread
template <typename P, typename S>
__global__ void final_accumulation_kernel(P* final_sums, P* final_results, unsigned nof_msms, unsigned nof_bms, unsigned c){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid>nof_msms) return;
  P final_result = P::zero();
  for (unsigned i = nof_bms; i >1; i--)
  {
    final_result = final_result + final_sums[i-1 + tid*nof_bms];  //add
    for (unsigned j=0; j<c; j++)  //double
    {
      final_result = final_result + final_result;
    }
  }
  final_results[tid] = final_result + final_sums[tid*nof_bms];

}

//this function computes msm using the bucket method
template <typename S, typename P, typename A>
void bucket_method_msm(unsigned bitsize, unsigned c, S *scalars, A *points, unsigned size, P* final_result, bool on_device, hipStream_t stream) {
  
  // std::cout<<"points"<<std::endl;
  // for (int i = 0; i < size; i++)
  // {
  //   std::cout<<points[i]<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"scalars"<<std::endl;
  // for (int i = 0; i < size; i++)
  // {
  //   std::cout<<scalars[i]<<" ";
  // }
  // std::cout<<std::endl;

  S *d_scalars;
  A *d_points;
  if (!on_device) {
    //copy scalars and point to gpu
    hipMallocAsync(&d_scalars, sizeof(S) * size, stream);
    hipMallocAsync(&d_points, sizeof(A) * size, stream);
    hipMemcpyAsync(d_scalars, scalars, sizeof(S) * size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_points, points, sizeof(A) * size, hipMemcpyHostToDevice, stream);
  }
  else {
    d_scalars = scalars;
    d_points = points;
  }

  P *buckets;
  //compute number of bucket modules and number of buckets in each module
  unsigned nof_bms = bitsize/c;
  unsigned msm_log_size = ceil(log2(size));
  unsigned bm_bitsize = ceil(log2(nof_bms));
  if (bitsize%c){
    nof_bms++;
  }
  unsigned top_bm_nof_missing_bits = c*nof_bms - bitsize;
  std::cout << "top_bm_nof_missing_bits" << top_bm_nof_missing_bits <<std::endl;
  // unsigned nof_buckets = nof_bms<<c;
  unsigned nof_buckets = nof_bms*((1<<(c-1))+1); //signed digits
  hipMalloc(&buckets, sizeof(P) * nof_buckets);

  // launch the bucket initialization kernel with maximum threads
  unsigned NUM_THREADS = 1 << 10;
  unsigned NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, nof_buckets);
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());

  unsigned *bucket_indices;
  unsigned *point_indices;
  hipMallocAsync(&bucket_indices, sizeof(unsigned) * size * (nof_bms+1), stream);
  hipMallocAsync(&point_indices, sizeof(unsigned) * size * (nof_bms+1), stream);

  //split scalars into digits
  NUM_THREADS = 1 << 10;
  NUM_BLOCKS = (size * (nof_bms+1) + NUM_THREADS - 1) / NUM_THREADS;
<<<<<<< HEAD
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(bucket_indices + size, point_indices + size, d_scalars, size, msm_log_size, 
                                                    nof_bms, bm_bitsize, c); //+size - leaving the first bm free for the out of place sort later
  
=======
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(bucket_indices + size, point_indices + size, d_scalars, size, msm_log_size, 
                                                    nof_bms, bm_bitsize, c, top_bm_nof_missing_bits); //+size - leaving the first bm free for the out of place sort later
                                                    hipDeviceSynchronize();
                                                    printf("cuda error %u",hipGetLastError());


  // hipDeviceSynchronize();
  // std::vector<unsigned> h_bucket_ind;
  // std::vector<unsigned> h_point_ind;
  // h_bucket_ind.reserve(size * (nof_bms+1));
  // h_point_ind.reserve(size * (nof_bms+1));
  // hipMemcpy(h_bucket_ind.data(), bucket_indices, sizeof(unsigned) * size * (nof_bms+1), hipMemcpyDeviceToHost);
  // hipMemcpy(h_point_ind.data(), point_indices, sizeof(unsigned) * size * (nof_bms+1), hipMemcpyDeviceToHost);
  //   std::cout<<hipGetLastError()<<std::endl;
  // std::cout<<"buckets inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_bucket_ind[i]<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"points inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_point_ind[i]<<" ";
  // }
  // std::cout<<std::endl;

  // std::cout<<"pure buckets inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_bucket_ind[i]%(1<<(c-1))<<" ";
  // }
  // std::cout<<std::endl;
  // std::cout<<"pure points inds"<<std::endl;
  // for (int i = 0; i < size * (nof_bms+1); i++)
  // {
  //   std::cout<<h_point_ind[i]%(1<<31)<<" ";
  // }
  // std::cout<<std::endl;
                                                    

>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  //sort indices - the indices are sorted from smallest to largest in order to group together the points that belong to each bucket
  unsigned *sort_indices_temp_storage{};
  size_t sort_indices_temp_storage_bytes;
  // The second to last parameter is the default value supplied explicitly to allow passing the stream
  // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for more info
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + size, bucket_indices,
<<<<<<< HEAD
                                 point_indices + size, point_indices, size, 0, sizeof(unsigned) * 8, stream);
  hipMallocAsync(&sort_indices_temp_storage, sort_indices_temp_storage_bytes, stream);
=======
                                 point_indices + size, point_indices, size);

  hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  for (unsigned i = 0; i < nof_bms; i++) {
    unsigned offset_out = i * size;
    unsigned offset_in = offset_out + size;
    // The second to last parameter is the default value supplied explicitly to allow passing the stream
    // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for more info
    hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + offset_in, bucket_indices + offset_out,
                                 point_indices + offset_in, point_indices + offset_out, size, 0, sizeof(unsigned) * 8, stream);
  }
  hipFreeAsync(sort_indices_temp_storage, stream);

  //find bucket_sizes
  unsigned *single_bucket_indices;
  unsigned *bucket_sizes;
  unsigned *nof_buckets_to_compute;
  hipMallocAsync(&single_bucket_indices, sizeof(unsigned)*nof_buckets, stream);
  hipMallocAsync(&bucket_sizes, sizeof(unsigned)*nof_buckets, stream);
  hipMallocAsync(&nof_buckets_to_compute, sizeof(unsigned), stream);
  unsigned *encode_temp_storage{};
  size_t encode_temp_storage_bytes = 0;
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size, stream);
  hipMallocAsync(&encode_temp_storage, encode_temp_storage_bytes, stream);
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, bucket_indices, single_bucket_indices, bucket_sizes,
                                        nof_buckets_to_compute, nof_bms*size, stream);
  hipFreeAsync(encode_temp_storage, stream);

  //get offsets - where does each new bucket begin
  unsigned* bucket_offsets;
  hipMallocAsync(&bucket_offsets, sizeof(unsigned)*nof_buckets, stream);
  unsigned* offsets_temp_storage{};
  size_t offsets_temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets, stream);
  hipMallocAsync(&offsets_temp_storage, offsets_temp_storage_bytes, stream);
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, nof_buckets, stream);
  hipFreeAsync(offsets_temp_storage, stream);

  //sort by bucket sizes
  unsigned* sorted_bucket_sizes;
  unsigned* sorted_bucket_offsets;
  unsigned* sorted_single_bucket_indices;
  hipMalloc(&sorted_bucket_sizes, sizeof(unsigned)*nof_buckets);
  hipMalloc(&sorted_bucket_offsets, sizeof(unsigned)*nof_buckets);
  hipMalloc(&sorted_single_bucket_indices, sizeof(unsigned)*nof_buckets);
  unsigned* sort_offsets_temp_storage{};
  size_t sort_offsets_temp_storage_bytes = 0;
  unsigned* sort_single_temp_storage{};
  size_t sort_single_temp_storage_bytes = 0;
  hipcub::DeviceRadixSort::SortPairsDescending(sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, bucket_offsets, sorted_bucket_offsets, nof_buckets);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, single_bucket_indices, sorted_single_bucket_indices, nof_buckets);
  hipMalloc(&sort_offsets_temp_storage, sort_offsets_temp_storage_bytes);
  hipMalloc(&sort_single_temp_storage, sort_single_temp_storage_bytes);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_offsets_temp_storage, sort_offsets_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, bucket_offsets, sorted_bucket_offsets, nof_buckets);
  hipcub::DeviceRadixSort::SortPairsDescending(sort_single_temp_storage, sort_single_temp_storage_bytes, bucket_sizes,
    sorted_bucket_sizes, single_bucket_indices, sorted_single_bucket_indices, nof_buckets);
  hipFree(sort_offsets_temp_storage);
  hipFree(sort_single_temp_storage);
  

  //launch the accumulation kernel with maximum threads
  NUM_THREADS = 1 << 8;
  // NUM_THREADS = 1 << 5;
  NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
<<<<<<< HEAD
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, 
                                                         d_points, nof_buckets, nof_buckets_to_compute, c+bm_bitsize);
=======
  // accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, point_indices, 
                                                        //  d_points, nof_buckets, nof_buckets_to_compute, c+bm_bitsize);                                              
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, sorted_bucket_offsets, sorted_bucket_sizes, sorted_single_bucket_indices, point_indices, 
                                                         d_points, nof_buckets, nof_buckets_to_compute, c+bm_bitsize, c);                   
  // accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, sorted_bucket_offsets, sorted_bucket_sizes, sorted_single_bucket_indices, point_indices, 
  //                                                        d_points, nof_buckets, nof_buckets_to_compute, c-1+bm_bitsize);                                              
                                                         hipDeviceSynchronize();
                                                         printf("cuda error %u",hipGetLastError());
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c

//   hipDeviceSynchronize();
// std::vector<P> h_buckets;
//   h_buckets.reserve(nof_buckets);
//     hipMemcpy(h_buckets.data(), buckets, sizeof(P) * nof_buckets, hipMemcpyDeviceToHost);
//     std::cout<<"buckets accumulated"<<std::endl;
//     for (unsigned i = 0; i < nof_buckets; i++)
//     {
//       std::cout<<h_buckets[i]<<" ";
//     }
//     std::cout<<std::endl;
  #ifdef SSM_SUM
    //sum each bucket
    NUM_THREADS = 1 << 10;
    NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
    ssm_buckets_kernel<fake_point, fake_scalar><<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, single_bucket_indices, nof_buckets, c);
   
    //sum each bucket module
    P* final_results;
    hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream);
    NUM_THREADS = 1<<c;
    NUM_BLOCKS = nof_bms;
    sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS, 0, stream>>>(buckets, final_results);
  #endif

  #ifdef BIG_TRIANGLE
    P* final_results;
    hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream);
    //launch the bucket module sum kernel - a thread for each bucket module
    NUM_THREADS = nof_bms;
    NUM_BLOCKS = 1;
<<<<<<< HEAD
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, final_results, nof_bms, c);
=======
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, final_results, nof_bms, c-1); //sighed digits
    // big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, final_results, nof_bms, c); 
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  #endif
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());

  // hipDeviceSynchronize();
  //   std::vector<P> h_final_results;
  //   h_final_results.reserve(nof_bms);
  //   hipMemcpy(h_final_results.data(), final_results, sizeof(P) * nof_bms, hipMemcpyDeviceToHost);
  //   std::cout<<"buckets summed"<<std::endl;
  //   for (unsigned i = 0; i < nof_bms; i++)
  //   {
  //     std::cout<<h_final_results[i]<<" ";
  //   }
  //   std::cout<<std::endl;


  P* d_final_result;
  if (!on_device)
    hipMallocAsync(&d_final_result, sizeof(P), stream);

  //launch the double and add kernel, a single thread
<<<<<<< HEAD
  final_accumulation_kernel<P, S><<<1,1,0,stream>>>(final_results, on_device ? final_result : d_final_result, 1, nof_bms, c);
  
=======
  final_accumulation_kernel<P, S><<<1,1>>>(final_results, on_device ? final_result : d_final_result, 1, nof_bms, c);
  hipDeviceSynchronize();
  printf("cuda error %u",hipGetLastError());
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  //copy final result to host
  hipStreamSynchronize(stream);
  if (!on_device)
    hipMemcpyAsync(final_result, d_final_result, sizeof(P), hipMemcpyDeviceToHost, stream);

  //free memory
  if (!on_device) {
    hipFreeAsync(d_points, stream);
    hipFreeAsync(d_scalars, stream);
    hipFreeAsync(d_final_result, stream);
  }
<<<<<<< HEAD
  hipFreeAsync(buckets, stream);
  hipFreeAsync(bucket_indices, stream);
  hipFreeAsync(point_indices, stream);
  hipFreeAsync(single_bucket_indices, stream);
  hipFreeAsync(bucket_sizes, stream);
  hipFreeAsync(nof_buckets_to_compute, stream);
  hipFreeAsync(bucket_offsets, stream);
  hipFreeAsync(final_results, stream);

  std::cout<< hipGetLastError() <<std::endl;
  hipStreamSynchronize(stream);
=======
  hipFree(buckets);
  hipFree(bucket_indices);
  hipFree(point_indices);
  hipFree(single_bucket_indices);
  hipFree(bucket_sizes);
  hipFree(nof_buckets_to_compute);
  hipFree(bucket_offsets);
  hipFree(sorted_bucket_sizes);
  hipFree(sorted_bucket_offsets);
  hipFree(sorted_single_bucket_indices);
  hipFree(final_results);
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
}

//this function computes msm using the bucket method
template <typename S, typename P, typename A>
void batched_bucket_method_msm(unsigned bitsize, unsigned c, S *scalars, A *points, unsigned batch_size, unsigned msm_size, P* final_results, bool on_device, hipStream_t stream){

  unsigned total_size = batch_size * msm_size;
  S *d_scalars;
  A *d_points;
  if (!on_device) {
    //copy scalars and point to gpu
    hipMallocAsync(&d_scalars, sizeof(S) * total_size, stream);
    hipMallocAsync(&d_points, sizeof(A) * total_size, stream);
    hipMemcpyAsync(d_scalars, scalars, sizeof(S) * total_size, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_points, points, sizeof(A) * total_size, hipMemcpyHostToDevice, stream);
  }
  else {
    d_scalars = scalars;
    d_points = points;
  }

  P *buckets;
  //compute number of bucket modules and number of buckets in each module
  unsigned nof_bms = bitsize/c;
  if (bitsize%c){
    nof_bms++;
  }
  unsigned msm_log_size = ceil(log2(msm_size));
  unsigned bm_bitsize = ceil(log2(nof_bms));
  unsigned nof_buckets = (nof_bms<<c);
  unsigned total_nof_buckets = nof_buckets*batch_size;
  hipMallocAsync(&buckets, sizeof(P) * total_nof_buckets, stream); 

  //lanch the bucket initialization kernel with maximum threads
  unsigned NUM_THREADS = 1 << 10;
  unsigned NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
  initialize_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, total_nof_buckets); 

  unsigned *bucket_indices;
  unsigned *point_indices;
  hipMallocAsync(&bucket_indices, sizeof(unsigned) * (total_size * nof_bms + msm_size), stream);
  hipMallocAsync(&point_indices, sizeof(unsigned) * (total_size * nof_bms + msm_size), stream);

  //split scalars into digits
  NUM_THREADS = 1 << 8;
  NUM_BLOCKS = (total_size * nof_bms + msm_size + NUM_THREADS - 1) / NUM_THREADS;
  split_scalars_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(bucket_indices + msm_size, point_indices + msm_size, d_scalars, total_size, 
                                                    msm_log_size, nof_bms, bm_bitsize, c); //+size - leaving the first bm free for the out of place sort later

  //sort indices - the indices are sorted from smallest to largest in order to group together the points that belong to each bucket
  unsigned *sorted_bucket_indices;
  unsigned *sorted_point_indices;
  hipMallocAsync(&sorted_bucket_indices, sizeof(unsigned) * (total_size * nof_bms), stream);
  hipMallocAsync(&sorted_point_indices, sizeof(unsigned) * (total_size * nof_bms), stream);

  unsigned *sort_indices_temp_storage{};
  size_t sort_indices_temp_storage_bytes;
  // The second to last parameter is the default value supplied explicitly to allow passing the stream
  // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for more info
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + msm_size, sorted_bucket_indices,
<<<<<<< HEAD
                                 point_indices + msm_size, sorted_point_indices, total_size * nof_bms, 0, sizeof(unsigned)*8, stream);
  hipMallocAsync(&sort_indices_temp_storage, sort_indices_temp_storage_bytes, stream);
  // The second to last parameter is the default value supplied explicitly to allow passing the stream
  // See https://nvlabs.github.io/cub/structcub_1_1_device_radix_sort.html#a65e82152de448c6373ed9563aaf8af7e for more info
=======
                                 point_indices + msm_size, sorted_point_indices, total_size * nof_bms);
  hipMalloc(&sort_indices_temp_storage, sort_indices_temp_storage_bytes);
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  hipcub::DeviceRadixSort::SortPairs(sort_indices_temp_storage, sort_indices_temp_storage_bytes, bucket_indices + msm_size, sorted_bucket_indices,
                                 point_indices + msm_size, sorted_point_indices, total_size * nof_bms, 0, sizeof(unsigned)*8, stream);
  hipFreeAsync(sort_indices_temp_storage, stream);

  //find bucket_sizes
  unsigned *single_bucket_indices;
  unsigned *bucket_sizes;
  unsigned *total_nof_buckets_to_compute;
  hipMallocAsync(&single_bucket_indices, sizeof(unsigned)*total_nof_buckets, stream);
  hipMallocAsync(&bucket_sizes, sizeof(unsigned)*total_nof_buckets, stream);
  hipMallocAsync(&total_nof_buckets_to_compute, sizeof(unsigned), stream);
  unsigned *encode_temp_storage{};
  size_t encode_temp_storage_bytes = 0;
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
                                        total_nof_buckets_to_compute, nof_bms*total_size, stream);  
  hipMallocAsync(&encode_temp_storage, encode_temp_storage_bytes, stream);
  hipcub::DeviceRunLengthEncode::Encode(encode_temp_storage, encode_temp_storage_bytes, sorted_bucket_indices, single_bucket_indices, bucket_sizes,
                                        total_nof_buckets_to_compute, nof_bms*total_size, stream);
  hipFreeAsync(encode_temp_storage, stream);

  //get offsets - where does each new bucket begin
  unsigned* bucket_offsets;
  hipMallocAsync(&bucket_offsets, sizeof(unsigned)*total_nof_buckets, stream);
  unsigned* offsets_temp_storage{};
  size_t offsets_temp_storage_bytes = 0;
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets, stream);
  hipMallocAsync(&offsets_temp_storage, offsets_temp_storage_bytes, stream);
  hipcub::DeviceScan::ExclusiveSum(offsets_temp_storage, offsets_temp_storage_bytes, bucket_sizes, bucket_offsets, total_nof_buckets, stream);
  hipFreeAsync(offsets_temp_storage, stream);

  //launch the accumulation kernel with maximum threads
  NUM_THREADS = 1 << 8;
  NUM_BLOCKS = (total_nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
<<<<<<< HEAD
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, sorted_point_indices,
=======
  accumulate_buckets_kernel<<<NUM_BLOCKS, NUM_THREADS>>>(buckets, bucket_offsets, bucket_sizes, single_bucket_indices, sorted_point_indices,
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
                                                        d_points, nof_buckets, total_nof_buckets_to_compute, c+bm_bitsize);

  #ifdef SSM_SUM
    //sum each bucket
    NUM_THREADS = 1 << 10;
    NUM_BLOCKS = (nof_buckets + NUM_THREADS - 1) / NUM_THREADS;
    ssm_buckets_kernel<P, S><<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, single_bucket_indices, nof_buckets, c);
   
    //sum each bucket module
    P* final_results;
    hipMallocAsync(&final_results, sizeof(P) * nof_bms, stream);
    NUM_THREADS = 1<<c;
    NUM_BLOCKS = nof_bms;
    sum_reduction_kernel<<<NUM_BLOCKS,NUM_THREADS, 0, stream>>>(buckets, final_results);
  #endif

  #ifdef BIG_TRIANGLE
    P* bm_sums;
    hipMallocAsync(&bm_sums, sizeof(P) * nof_bms * batch_size, stream);
    //launch the bucket module sum kernel - a thread for each bucket module
    NUM_THREADS = 1<<8;
    NUM_BLOCKS = (nof_bms*batch_size + NUM_THREADS - 1) / NUM_THREADS;
    big_triangle_sum_kernel<<<NUM_BLOCKS, NUM_THREADS, 0, stream>>>(buckets, bm_sums, nof_bms*batch_size, c);
  #endif

  P* d_final_results;
  if (!on_device)
    hipMallocAsync(&d_final_results, sizeof(P)*batch_size, stream);

  //launch the double and add kernel, a single thread for each msm
  NUM_THREADS = 1<<8;
  NUM_BLOCKS = (batch_size + NUM_THREADS - 1) / NUM_THREADS;
<<<<<<< HEAD
  final_accumulation_kernel<P, S><<<NUM_BLOCKS,NUM_THREADS, 0, stream>>>(bm_sums, on_device ? final_results : d_final_results, batch_size, nof_bms, c);
  
=======
  final_accumulation_kernel<P, S><<<NUM_BLOCKS,NUM_THREADS>>>(bm_sums, on_device ? final_results : d_final_results, batch_size, nof_bms, c);

>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  //copy final result to host
  if (!on_device)
    hipMemcpyAsync(final_results, d_final_results, sizeof(P)*batch_size, hipMemcpyDeviceToHost, stream);

  //free memory
  if (!on_device) {
    hipFreeAsync(d_points, stream);
    hipFreeAsync(d_scalars, stream);
    hipFreeAsync(d_final_results, stream);
  }
  hipFreeAsync(buckets, stream);
  hipFreeAsync(bucket_indices, stream);
  hipFreeAsync(point_indices, stream);
  hipFreeAsync(sorted_bucket_indices, stream);
  hipFreeAsync(sorted_point_indices, stream);
  hipFreeAsync(single_bucket_indices, stream);
  hipFreeAsync(bucket_sizes, stream);
  hipFreeAsync(total_nof_buckets_to_compute, stream);
  hipFreeAsync(bucket_offsets, stream);
  hipFreeAsync(bm_sums, stream);

  hipStreamSynchronize(stream);
}


//this kernel converts affine points to projective points
//each thread deals with a single point
template <typename P, typename A>
__global__ void to_proj_kernel(A* affine_points, P* proj_points, unsigned N){
  
  unsigned tid = (blockIdx.x * blockDim.x) + threadIdx.x;
  if (tid < N) proj_points[tid] = P::from_affine(affine_points[tid]);
}

//the function computes msm using ssm
template <typename S, typename P, typename A>
<<<<<<< HEAD
void short_msm(S *h_scalars, A *h_points, unsigned size, P* h_final_result, hipStream_t stream){ //works up to 2^8
=======
void short_msm(S *h_scalars, A *h_points, unsigned size, P* h_final_result){ //works up to 2^8
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  S *scalars;
  A *a_points;
  P *p_points;
  P *results;

  hipMallocAsync(&scalars, sizeof(S) * size, stream);
  hipMallocAsync(&a_points, sizeof(A) * size, stream);
  hipMallocAsync(&p_points, sizeof(P) * size, stream);
  hipMallocAsync(&results, sizeof(P) * size, stream);

  //copy inputs to device
  hipMemcpyAsync(scalars, h_scalars, sizeof(S) * size, hipMemcpyHostToDevice, stream);
  hipMemcpyAsync(a_points, h_points, sizeof(A) * size, hipMemcpyHostToDevice, stream);

  //convert to projective representation and multiply each point by its scalar using single scalar multiplication
  unsigned NUM_THREADS = size;
  to_proj_kernel<<<1,NUM_THREADS, 0, stream>>>(a_points, p_points, size);
  ssm_kernel<<<1,NUM_THREADS, 0, stream>>>(scalars, p_points, results, size);

  P *final_result;
  hipMallocAsync(&final_result, sizeof(P), stream);

  //assuming msm size is a power of 2
  //sum all the ssm results
  NUM_THREADS = size;
  sum_reduction_kernel<<<1,NUM_THREADS, 0, stream>>>(results, final_result);

  //copy result to host
  hipStreamSynchronize(stream);
  hipMemcpyAsync(h_final_result, final_result, sizeof(P), hipMemcpyDeviceToHost, stream);

  //free memory
  hipFreeAsync(scalars, stream);
  hipFreeAsync(a_points, stream);
  hipFreeAsync(p_points, stream);
  hipFreeAsync(results, stream);
  hipFreeAsync(final_result, stream);

}

//the function computes msm on the host using the naive method
template <typename A, typename S, typename P>
void reference_msm(S* scalars, A* a_points, unsigned size){
  
  P *points = new P[size];
  // P points[size];
  for (unsigned i = 0; i < size ; i++)
  {
    points[i] = P::from_affine(a_points[i]);
  }

  P res = P::zero();
  
  for (unsigned i = 0; i < size; i++)
  {
    res = res + scalars[i]*points[i];
  }

  std::cout<<"reference results"<<std::endl;
  std::cout<<P::to_affine(res)<<std::endl;
  
}

unsigned get_optimal_c(const unsigned size) {
  if (size < 17)
    return 1;
<<<<<<< HEAD
  // return 15;
=======
  // return 17;
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
  return ceil(log2(size))-4;
}

//this function is used to compute msms of size larger than 256
template <typename S, typename P, typename A>
void large_msm(S* scalars, A* points, unsigned size, P* result, bool on_device, hipStream_t stream){
  unsigned c = get_optimal_c(size);
  // unsigned c = 4;
  // unsigned bitsize = 32;
<<<<<<< HEAD
  unsigned bitsize = 255;
  bucket_method_msm(bitsize, c, scalars, points, size, result, on_device, stream);
=======
  unsigned bitsize = 253; //get from field
  bucket_method_msm(bitsize, c, scalars, points, size, result, on_device);
>>>>>>> ed9de3d1e982ef04dd4b84c9912d3ee68ebfd52c
}

// this function is used to compute a batches of msms of size larger than 256
template <typename S, typename P, typename A>
void batched_large_msm(S* scalars, A* points, unsigned batch_size, unsigned msm_size, P* result, bool on_device, hipStream_t stream){
  unsigned c = get_optimal_c(msm_size);
  // unsigned c = 6;
  // unsigned bitsize = 32;
  unsigned bitsize = 255;
  batched_bucket_method_msm(bitsize, c, scalars, points, batch_size, msm_size, result, on_device, stream);
}
#endif
